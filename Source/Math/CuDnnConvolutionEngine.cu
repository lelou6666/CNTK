#include "hip/hip_runtime.h"
//
// Copyright (c) Microsoft. All rights reserved.
// Licensed under the MIT license. See LICENSE.md file in the project root for full license information.
//

#include "stdafx.h"
#include "CuDnnConvolutionEngine.h"
#include "GPUMatrix.h"
#ifdef USE_CUDNN
#include <hipDNN.h>
#include "CuDnnConvolutionEngine.cuh"

template <>
const char* CudaErrString<hipdnnStatus_t>(hipdnnStatus_t x)
{
    return hipdnnGetErrorString(x);
}

// A note on the formats: CNTK originally used NHWC for input/output tensors and CHWN for filters.
// Such formats have very limited support in cuDNN and not used in other frameworks.
// CNTK with cuDNN by default uses NCHW formats for both inputs/outputs and filters.
#define TENSOR_FORMAT HIPDNN_TENSOR_NCHW
#define FILTER_FORMAT HIPDNN_TENSOR_NCHW
#endif

namespace Microsoft { namespace MSR { namespace CNTK {

template <class ElemType>
bool CuDnnConvolutionEngineFactory<ElemType>::IsSupported(DEVICEID_TYPE deviceId)
{
// REVIEW alexeyk: compile-time for now, make runtime, config-driven.
#ifdef USE_CUDNN
    hipDeviceProp_t props = {0};
    return hipGetDeviceProperties(&props, deviceId) == hipSuccess && props.major >= 3;
#else
    UNUSED(deviceId);
    return false;
#endif
}

CudaTimer::~CudaTimer()
{
    // TODO: Should not throw if std::uncaught_exception()
    if (m_start != nullptr)
        CUDA_CALL(hipEventDestroy(reinterpret_cast<hipEvent_t>(m_start)));
    if (m_stop != nullptr)
        CUDA_CALL(hipEventDestroy(reinterpret_cast<hipEvent_t>(m_stop)));
}
void CudaTimer::Start()
{
    hipEvent_t start;
    hipEvent_t stop;
    if (m_start != nullptr)
        CUDA_CALL(hipEventDestroy(reinterpret_cast<hipEvent_t>(m_start)));
    if (m_stop != nullptr)
        CUDA_CALL(hipEventDestroy(reinterpret_cast<hipEvent_t>(m_stop)));
    CUDA_CALL(hipEventCreate(&start));
    CUDA_CALL(hipEventCreate(&stop));
    m_start = start;
    m_stop = stop;
    CUDA_CALL(hipEventRecord(start, GetStream()));
}
void CudaTimer::Stop()
{
    CUDA_CALL(hipEventRecord(reinterpret_cast<hipEvent_t>(m_stop), GetStream()));
    CUDA_CALL(hipEventSynchronize(reinterpret_cast<hipEvent_t>(m_stop)));
}
float CudaTimer::Elapsed()
{
    float ms;
    CUDA_CALL(hipEventElapsedTime(&ms, reinterpret_cast<hipEvent_t>(m_start), reinterpret_cast<hipEvent_t>(m_stop)));
    return ms;
}

#ifdef USE_CUDNN

static bool IsGpu(DEVICEID_TYPE deviceId)
{
    return deviceId >= 0;
}

class CuDnnTensor4D : public ConvolutionTensor4D
{
public:
    CuDnnTensor4D(size_t w, size_t h, size_t c, size_t n, hipdnnDataType_t dataType)
        : ConvolutionTensor4D(w, h, c, n), m_dataType(dataType), m_tensor(nullptr)
    {
        CUDNN_CALL(hipdnnCreateTensorDescriptor(&m_tensor));
        CUDNN_CALL(hipdnnSetTensor4dDescriptor(m_tensor, TENSOR_FORMAT, dataType,
                                              static_cast<int>(n), static_cast<int>(c), static_cast<int>(h), static_cast<int>(w)));
    }

public:
    operator hipdnnTensorDescriptor_t() const
    {
        return m_tensor;
    }

    ~CuDnnTensor4D() noexcept
    {
        if (m_tensor != nullptr)
        {
            // TODO: Check for error code and throw if !std::uncaught_exception()
            hipdnnDestroyTensorDescriptor(m_tensor);
            m_tensor = nullptr;
        }
    }

    void setN(size_t newN) override
    {
        ConvolutionTensor4D::setN(newN);
        CUDNN_CALL(hipdnnSetTensor4dDescriptor(m_tensor, TENSOR_FORMAT, m_dataType,
                                              static_cast<int>(n()), static_cast<int>(c()), static_cast<int>(h()), static_cast<int>(w())));
    }

private:
    hipdnnDataType_t m_dataType;
    hipdnnTensorDescriptor_t m_tensor;
};

class CuDnnFilter : public ConvolutionFilter
{
public:
    CuDnnFilter(size_t w, size_t h, size_t c, size_t k, hipdnnDataType_t dataType)
        : ConvolutionFilter(w, h, c, k), m_filter(nullptr)
    {
        CUDNN_CALL(hipdnnCreateFilterDescriptor(&m_filter));
        CUDNN_CALL(cudnnSetFilter4dDescriptor_v4(m_filter, dataType, FILTER_FORMAT,
                                                 static_cast<int>(k), static_cast<int>(c), static_cast<int>(h), static_cast<int>(w)));
    }

public:
    operator hipdnnFilterDescriptor_t() const
    {
        return m_filter;
    }

    ~CuDnnFilter() noexcept
    {
        if (m_filter != nullptr)
        {
            // TODO: Check for error code and throw if !std::uncaught_exception()
            hipdnnDestroyFilterDescriptor(m_filter);
            m_filter = nullptr;
        }
    }

private:
    hipdnnFilterDescriptor_t m_filter;
};

class CuDnnConvolutionDescriptor : public ConvolutionDescriptor
{
public:
    CuDnnConvolutionDescriptor(size_t wStride, size_t hStride, size_t wPad, size_t hPad)
        : ConvolutionDescriptor(wStride, hStride, wPad > 0 || hPad > 0), m_conv(nullptr)
    {
        CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&m_conv));
        CUDNN_CALL(hipdnnSetConvolution2dDescriptor(m_conv,
                                                   static_cast<int>(hPad), static_cast<int>(wPad),
                                                   static_cast<int>(hStride), static_cast<int>(wStride),
                                                   1, 1, HIPDNN_CROSS_CORRELATION));
    }

public:
    operator hipdnnConvolutionDescriptor_t() const
    {
        return m_conv;
    }

    ~CuDnnConvolutionDescriptor() noexcept
    {
        if (m_conv != nullptr)
        {
            // TODO: Check for error code and throw if !std::uncaught_exception()
            hipdnnDestroyConvolutionDescriptor(m_conv);
            m_conv = nullptr;
        }
    }

private:
    hipdnnConvolutionDescriptor_t m_conv;
};

class CuDnnPoolingDescriptor : public PoolingDescriptor
{
public:
    CuDnnPoolingDescriptor(PoolKind kind, size_t w, size_t h, size_t wStride, size_t hStride, size_t wPad, size_t hPad)
        : PoolingDescriptor(kind, w, h, wStride, hStride, wPad, hPad), m_pool(nullptr)
    {
        assert(kind == PoolKind::Max || kind == PoolKind::Average);

        CUDNN_CALL(hipdnnCreatePoolingDescriptor(&m_pool));
        CUDNN_CALL(hipdnnSetPooling2dDescriptor(m_pool,
                                               kind == PoolKind::Max ? HIPDNN_POOLING_MAX : HIPDNN_POOLING_AVERAGE_COUNT_EXCLUDE_PADDING,
                                               static_cast<int>(h), static_cast<int>(w),
                                               static_cast<int>(hPad), static_cast<int>(wPad),
                                               static_cast<int>(hStride), static_cast<int>(wStride)));
    }

public:
    operator hipdnnPoolingDescriptor_t() const
    {
        return m_pool;
    }

    ~CuDnnPoolingDescriptor() noexcept
    {
        if (m_pool != nullptr)
        {
            // TODO: Check for error code and throw if !std::uncaught_exception()
            hipdnnDestroyPoolingDescriptor(m_pool);
            m_pool = nullptr;
        }
    }

private:
    hipdnnPoolingDescriptor_t m_pool;
};

template <typename CuDnnT, typename In>
static CuDnnT& As(In& src)
{
    // Do dynamic_cast only in debug builds and static_cast in release builds.
    assert(dynamic_cast<CuDnnT*>(&src) != nullptr);
    return static_cast<CuDnnT&>(src);
}
static const CuDnnTensor4D& t(const ConvolutionTensor4D& src)
{
    return As<const CuDnnTensor4D>(src);
}
static const CuDnnFilter& f(const ConvolutionFilter& src)
{
    return As<const CuDnnFilter>(src);
}
static const CuDnnConvolutionDescriptor& cd(const ConvolutionDescriptor& src)
{
    return As<const CuDnnConvolutionDescriptor>(src);
}
static const CuDnnPoolingDescriptor& p(const PoolingDescriptor& src)
{
    return As<const CuDnnPoolingDescriptor>(src);
}
template <typename ElemType>
static ElemType* ptr(Matrix<ElemType>& src)
{
    return src.BufferPointer();
}
template <typename ElemType>
static const ElemType* ptr(const Matrix<ElemType>& src)
{
    return src.BufferPointer();
}

template <typename ElemType>
struct Consts
{
    static const ElemType Zero;
    static const ElemType One;
};
template <>
const float Consts<float>::One = 1;
template <>
const double Consts<double>::One = 1;
template <>
const float Consts<float>::Zero = 0;
template <>
const double Consts<double>::Zero = 0;

template <typename ElemType>
class CuDnnConvolutionEngine : public ConvolutionEngine<ElemType>
{
public:
    using Base = ConvolutionEngine<ElemType>;
    using typename Base::Mat;
    using typename Base::Tensor4D;
    using typename Base::Filter;
    using typename Base::ConvDesc;

    CuDnnConvolutionEngine(DEVICEID_TYPE deviceId, ImageLayoutKind imageLayout, size_t maxTempMemSizeInSamples, BatchNormImpl bnImpl)
        : Base(deviceId, imageLayout), m_maxTempMemSizeInSamples(maxTempMemSizeInSamples), m_bnImpl(bnImpl), m_stream(GetStream()), m_cudnn(nullptr)
    {
        CUDNN_CALL(hipdnnCreate(&m_cudnn));
        CUDNN_CALL(hipdnnSetStream(m_cudnn, m_stream));
    }

    ~CuDnnConvolutionEngine()
    {
        if (m_cudnn != nullptr)
        {
            // TODO: Check for error code and throw if !std::uncaught_exception()
            hipdnnDestroy(m_cudnn);
            m_cudnn = nullptr;
        }
    }

protected:
    using Base::m_deviceId;
    using Base::m_imageLayout;

    void EnsureCompatible() override
    {
        if (m_imageLayout != ImageLayoutKind::CHW)
            RuntimeError("cuDNN convolution engine supports only CHW/cudnn layout.");
        if (!IsGpu(m_deviceId))
            RuntimeError("cuDNN convolution engine supports GPU devices only.");
    }

    void ForwardCore(const Tensor4D& inT, const Mat& in, const Filter& filterT, const Mat& filter, const ConvDesc& convDesc,
                     const Tensor4D& outT, Mat& out, Mat& workspace) override
    {
        // Find best algo and allocate temp buffer, if needed.
        auto finder = [&](int& calgo, hipdnnConvolutionFwdAlgoPerf_t algoPerf[MaxAlgoCount]) -> hipdnnStatus_t
        {
            return hipdnnFindConvolutionForwardAlgorithm(m_cudnn, t(inT), f(filterT), cd(convDesc), t(outT), MaxAlgoCount, &calgo, algoPerf);
        };
        FindBestAlgo(t(inT), m_fwdAlgo, finder);
        if (m_fwdAlgo.Algo.memory > 0)
            workspace.Resize((m_fwdAlgo.Algo.memory + sizeof(ElemType) - 1) / sizeof(ElemType), 1);
        // Perform forward convolution operation.
        auto err = hipdnnConvolutionForward(m_cudnn, &C::One, t(inT), ptr(in), f(filterT), ptr(filter), cd(convDesc),
                                           m_fwdAlgo.Algo.algo, ptr(workspace), m_fwdAlgo.Algo.memory, &C::Zero, t(outT), ptr(out));
        // There might be a case where cuDNN fails due to workspace being too small, try using no-workspace algo instead.
        // REVIEW alexeyk: NVIDIA is currently reviewing this issue.
        if (HIPDNN_STATUS_INVALID_VALUE == err && m_fwdAlgo.Algo.memory > 0)
        {
            auto err2 = hipdnnConvolutionForward(m_cudnn, &C::One, t(inT), ptr(in), f(filterT), ptr(filter), cd(convDesc),
                                                m_fwdAlgo.NoWorkspaceAlgo, nullptr, 0, &C::Zero, t(outT), ptr(out));
            // Update original error in case of success.
            if (HIPDNN_STATUS_SUCCESS == err2)
                err = HIPDNN_STATUS_SUCCESS;
        }
        CUDNN_CALL(err);
    }

    void BackwardDataCore(const Tensor4D& srcGradT, const Mat& srcGrad, const Filter& filterT, const Mat& filter, const ConvDesc& convDesc,
                          const Tensor4D& gradT, Mat& grad, Mat& workspace) override
    {
        // Find best algo and allocate temp buffer, if needed.
        auto finder = [&](int& calgo, hipdnnConvolutionBwdDataAlgoPerf_t algoPerf[MaxAlgoCount]) -> hipdnnStatus_t
        {
            return hipdnnFindConvolutionBackwardDataAlgorithm(m_cudnn, f(filterT), t(srcGradT), cd(convDesc), t(gradT), MaxAlgoCount, &calgo, algoPerf);
        };
        FindBestAlgo(t(srcGradT), m_backDataAlgo, finder);
        if (m_backDataAlgo.Algo.memory > 0)
            workspace.Resize((m_backDataAlgo.Algo.memory + sizeof(ElemType) - 1) / sizeof(ElemType), 1);
        // Compute gradients with respect to the output tensor (data).
        CUDNN_CALL(hipdnnConvolutionBackwardData(m_cudnn, &C::One, f(filterT), ptr(filter), t(srcGradT), ptr(srcGrad), cd(convDesc), m_backDataAlgo.Algo.algo,
                                                ptr(workspace), m_backDataAlgo.Algo.memory, &C::One, t(gradT), ptr(grad)));
    }

    void BackwardFilterCore(const Tensor4D& srcGradT, const Mat& srcGrad, const Tensor4D& inT, const Mat& in, const ConvDesc& convDesc,
                            const Filter& filterT, Mat& filter, bool /*allowReuse*/, Mat& workspace) override
    {
        // Find best algo and allocate temp buffer, if needed.
        auto finder = [&](int& calgo, hipdnnConvolutionBwdFilterAlgoPerf_t algoPerf[MaxAlgoCount]) -> hipdnnStatus_t
        {
            return hipdnnFindConvolutionBackwardFilterAlgorithm(m_cudnn, t(inT), t(srcGradT), cd(convDesc), f(filterT), MaxAlgoCount, &calgo, algoPerf);
        };
        FindBestAlgo(t(inT), m_backFiltAlgo, finder);
        if (m_backFiltAlgo.Algo.memory > 0)
            workspace.Resize((m_backFiltAlgo.Algo.memory + sizeof(ElemType) - 1) / sizeof(ElemType), 1);
        // Compute gradients with respect to the output tensor (data).
        CUDNN_CALL(hipdnnConvolutionBackwardFilter(m_cudnn, &C::One, t(inT), ptr(in), t(srcGradT), ptr(srcGrad), cd(convDesc), m_backFiltAlgo.Algo.algo,
                                                  ptr(workspace), m_backFiltAlgo.Algo.memory, &C::One, f(filterT), ptr(filter)));
    }

    void EnsureCompatibleBatchNorm(bool spatial) override
    {
        if (!IsGpu(m_deviceId))
            InvalidArgument("cuDNN engine does not support batch normalization on CPUs.");
        if (spatial && m_imageLayout != ImageLayoutKind::CHW)
            InvalidArgument("cuDNN engine batch normalization currently supports only CHW data layout for convolutional nodes.");
    }

    void NormalizeBatchCore(const Tensor4D& inT, const Mat& in, const Tensor4D& scaleBiasT, const Mat& scale, const Mat& bias,
                            bool spatial, double expAvgFactor, Mat& runMean, Mat& runInvStdDev, Mat& out,
                            double epsilon, Mat& saveMean, Mat& saveInvStdDev) override
    {
        if (m_bnImpl == BatchNormImpl::CuDnn)
        {
            hipdnnBatchNormMode_t mode = spatial ? HIPDNN_BATCHNORM_SPATIAL : HIPDNN_BATCHNORM_PER_ACTIVATION;
            // cuDNN will fail with BAD_PARAM if epsilon < HIPDNN_BN_MIN_EPSILON.
            epsilon = std::max(epsilon, HIPDNN_BN_MIN_EPSILON);
            CUDNN_CALL(hipdnnBatchNormalizationForwardTraining(m_cudnn, mode, &C::One, &C::Zero, t(inT), ptr(in), t(inT), ptr(out),
                t(scaleBiasT), ptr(scale), ptr(bias), expAvgFactor, ptr(runMean), ptr(runInvStdDev), 
                epsilon, ptr(saveMean), ptr(saveInvStdDev)));
        }
        else if (m_bnImpl == BatchNormImpl::Cntk)
        {
            epsilon = std::max(epsilon, 1e-9);
            CUDA_CALL(BatchNormalizationForwardTraining(inT, spatial, ptr(in), ptr(out), ptr(scale), ptr(bias),
                                                        expAvgFactor, ptr(runMean), ptr(runInvStdDev),
                                                        epsilon, ptr(saveMean), ptr(saveInvStdDev), m_stream));
        }
        else
            RuntimeError("Provided batch norm implementation (%d) is not supported.", m_bnImpl);
    }

    void NormalizeBatchInferenceCore(const Tensor4D& inT, const Mat& in, const Tensor4D& scaleBiasT, const Mat& scale, const Mat& bias,
                                     bool spatial, const Mat& runMean, const Mat& runInvStdDev, Mat& out) override
    {
        if (m_bnImpl == BatchNormImpl::CuDnn)
        {
            hipdnnBatchNormMode_t mode = spatial ? HIPDNN_BATCHNORM_SPATIAL : HIPDNN_BATCHNORM_PER_ACTIVATION;
            CUDNN_CALL(hipdnnBatchNormalizationForwardInference(m_cudnn, mode, &C::One, &C::Zero, t(inT), ptr(in), t(inT), ptr(out),
                                                               t(scaleBiasT), ptr(scale), ptr(bias), ptr(runMean), ptr(runInvStdDev), HIPDNN_BN_MIN_EPSILON));
        }
        else if (m_bnImpl == BatchNormImpl::Cntk)
        {
            CUDA_CALL(BatchNormalizationForwardInference(inT, spatial, ptr(in), ptr(out), ptr(scale), ptr(bias),
                                                         ptr(runMean), ptr(runInvStdDev), m_stream));
        }
        else
            RuntimeError("Provided batch norm implementation (%d) is not supported.", m_bnImpl);
    }

    void BackwardNormalizeBatchCore(const Tensor4D& inT, const Mat& in, const Mat& srcGrad, Mat& grad,
                                    const Tensor4D& scaleBiasT, const Mat& scale, bool spatial, const Mat& saveMean, const Mat& saveInvStdDev,
                                    Mat& scaleGrad, Mat& biasGrad) override
    {
        if (m_bnImpl == BatchNormImpl::CuDnn)
        {
            hipdnnBatchNormMode_t mode = spatial ? HIPDNN_BATCHNORM_SPATIAL : HIPDNN_BATCHNORM_PER_ACTIVATION;
// REVIEW alexeyk: remove once Philly is upgraded to prod version.
#if CUDNN_PATCHLEVEL >= 7
            CUDNN_CALL(hipdnnBatchNormalizationBackward(m_cudnn, mode, &C::One, &C::One, &C::One, &C::One, t(inT), ptr(in), t(inT), ptr(srcGrad), t(inT), ptr(grad),
                                                       t(scaleBiasT), ptr(scale), ptr(scaleGrad), ptr(biasGrad), HIPDNN_BN_MIN_EPSILON, ptr(saveMean), ptr(saveInvStdDev)));
#else
            CUDNN_CALL(hipdnnBatchNormalizationBackward(m_cudnn, mode, &C::One, &C::One, t(inT), ptr(in), t(inT), ptr(srcGrad), t(inT), ptr(grad),
                t(scaleBiasT), ptr(scale), ptr(scaleGrad), ptr(biasGrad), HIPDNN_BN_MIN_EPSILON, ptr(saveMean), ptr(saveInvStdDev)));
#endif

        }
        else if (m_bnImpl == BatchNormImpl::Cntk)
        {
            CUDA_CALL(BatchNormalizationBackward(inT, spatial, ptr(in), ptr(srcGrad), ptr(grad), ptr(scale), ptr(scaleGrad), ptr(biasGrad),
                                                 ptr(saveMean), ptr(saveInvStdDev), m_stream));
        }
        else
            RuntimeError("Provided batch norm implementation (%d) is not supported.", m_bnImpl);
    }

private:
    static const int MaxAlgoCount = 10;

    template <typename TAlgo, typename TFinder>
    void FindBestAlgo(const CuDnnTensor4D& t, TAlgo& algo, TFinder finder)
    {
        if (!algo.NeedAutotuning(t))
            return;
        using CuDnnAlgoT = decltype(TAlgo::Algo);
        CuDnnAlgoT algoPerf[MaxAlgoCount];
        int calgo = 0;
        CUDNN_CALL(finder(calgo, algoPerf));
        assert(calgo > 0);
        size_t maxMem = m_maxTempMemSizeInSamples == 0 ? (std::numeric_limits<size_t>::max)() : t.w() * t.h() * t.c() * m_maxTempMemSizeInSamples * sizeof(ElemType);
        auto res = std::find_if(algoPerf, algoPerf + calgo,
            [=](const CuDnnAlgoT& cur)
            {
                return cur.status == HIPDNN_STATUS_SUCCESS && cur.memory <= maxMem;
            });
        if (res == algoPerf + calgo)
            RuntimeError("cuDNN could not find suitable algorithm for the current convolution configuration.");
        algo.CurMBSize = t.n();
        algo.Algo = *res;
        res = std::find_if(algoPerf, algoPerf + calgo,
            [](const CuDnnAlgoT& cur)
            {
                return cur.status == HIPDNN_STATUS_SUCCESS && cur.memory == 0;
            });
        if (res == algoPerf + calgo)
        {
            // In theory, this should never happen.
            RuntimeError("cuDNN could not find no-workspace algorithm for the current convolution configuration.");
        }
        else
            algo.NoWorkspaceAlgo = (*res).algo;
    }

private:
    template <typename T>
    struct ConvAlgoInfo
    {
        using CuDnnAlgoT = decltype(T::algo);

        ConvAlgoInfo()
            : CurMBSize(0)
        {
            Algo.status = HIPDNN_STATUS_NOT_INITIALIZED;
            NoWorkspaceAlgo = (CuDnnAlgoT)-1;
        }
        // Current mini-batch size, needed for re-computing statistics in auto-tuner.
        size_t CurMBSize;
        T Algo;
        CuDnnAlgoT NoWorkspaceAlgo;

        bool NeedAutotuning(const CuDnnTensor4D& t)
        {
            // Need to re-run auto-tuner in case minibatch size is increased.
            // If minibatch size is decreased we assume that previously selected algorithm requires less or the same amount of workspace.
            // This is done to avoid re-running auto-tuner every time in case minibatch size changes frequently (e.g. when distributed reading is enabled).
            // REVIEW alexeyk: potentially, this might cause some perf issues if better (faster) algo can be selected for a smaller mininbatch.
            // We also need to reset auto-tuning status at the beginning of each epoch but ComputationNode currently does not provide such notification.
            // We assume no other dimensions of tensors can change so we don't check it.
            // REVIEW alexeyk: review once we get response from NVIDIA.
            return (Algo.status != HIPDNN_STATUS_SUCCESS || t.n() > CurMBSize);
        }
    };

    using C = Consts<ElemType>;

    // REVIEW alexeyk: currently limit is set once in ctor though in CNTK it can be, theoretically, changed in runtime.
    size_t m_maxTempMemSizeInSamples;
    BatchNormImpl m_bnImpl;
    hipdnnHandle_t m_cudnn;
    hipStream_t m_stream;
    ConvAlgoInfo<hipdnnConvolutionFwdAlgoPerf_t> m_fwdAlgo;
    ConvAlgoInfo<hipdnnConvolutionBwdDataAlgoPerf_t> m_backDataAlgo;
    ConvAlgoInfo<hipdnnConvolutionBwdFilterAlgoPerf_t> m_backFiltAlgo;
};

template <class ElemType>
class CuDnnPoolingEngine : public PoolingEngine<ElemType>
{
public:
    using Base = PoolingEngine<ElemType>;
    using typename Base::Tensor4D;
    using typename Base::PoolDesc;
    using typename Base::Mat;

public:
    CuDnnPoolingEngine(DEVICEID_TYPE deviceId, ImageLayoutKind imageLayout)
        : Base(deviceId, imageLayout), m_cudnn(nullptr)
    {
        CUDNN_CALL(hipdnnCreate(&m_cudnn));
        CUDNN_CALL(hipdnnSetStream(m_cudnn, GetStream()));
    }

    ~CuDnnPoolingEngine()
    {
        if (m_cudnn != nullptr)
        {
            // TODO: Check for error code and throw if !std::uncaught_exception()
            hipdnnDestroy(m_cudnn);
            m_cudnn = nullptr;
        }
    }

protected:
    using Base::m_deviceId;
    using Base::m_imageLayout;

    void EnsureCompatible() override
    {
        if (m_imageLayout != ImageLayoutKind::CHW)
            RuntimeError("cuDNN pooling engine supports only CHW/cudnn layout.");
        if (!IsGpu(m_deviceId))
            RuntimeError("cuDNN pooling engine supports GPU devices only.");
    }

    void ForwardCore(const Tensor4D& inT, const Mat& in, const PoolDesc& poolDesc, const Tensor4D& outT, Mat& out) override
    {
        CUDNN_CALL(hipdnnPoolingForward(m_cudnn, p(poolDesc), &C::One, t(inT), ptr(in), &C::Zero, t(outT), ptr(out)));
    }

    void BackwardCore(const Tensor4D& outT, const Mat& out, const Mat& srcGrad, const PoolDesc& poolDesc, const Tensor4D& inT, const Mat& in, Mat& grad) override
    {
        CUDNN_CALL(hipdnnPoolingBackward(m_cudnn, p(poolDesc), &C::One, t(outT), ptr(out), t(outT), ptr(srcGrad),
                                        t(inT), ptr(in), &C::One, t(inT), ptr(grad)));
    }

private:
    using C = Consts<ElemType>;

    hipdnnHandle_t m_cudnn;
};

template <class ElemType>
typename CuDnnConvolutionEngineFactory<ElemType>::Tensor4DPtr CuDnnConvolutionEngineFactory<ElemType>::CreateTensor(size_t w, size_t h, size_t c, size_t n)
{
    // REVIEW alexeyk: assert fires in GCC but not in VC++.
    // static_assert(false, "cuDNN engine currently supports only single and double precision tensors.");
    RuntimeError("Not implemented.");
}
template <>
typename CuDnnConvolutionEngineFactory<float>::Tensor4DPtr CuDnnConvolutionEngineFactory<float>::CreateTensor(size_t w, size_t h, size_t c, size_t n)
{
    return std::make_unique<CuDnnTensor4D>(w, h, c, n, HIPDNN_DATA_FLOAT);
}
template <>
typename CuDnnConvolutionEngineFactory<double>::Tensor4DPtr CuDnnConvolutionEngineFactory<double>::CreateTensor(size_t w, size_t h, size_t c, size_t n)
{
    return std::make_unique<CuDnnTensor4D>(w, h, c, n, HIPDNN_DATA_DOUBLE);
}

template <class ElemType>
typename CuDnnConvolutionEngineFactory<ElemType>::FilterPtr CuDnnConvolutionEngineFactory<ElemType>::CreateFilter(size_t w, size_t h, size_t c, size_t k)
{
    // REVIEW alexeyk: assert fires in GCC but not in VC++.
    // static_assert(false, "cuDNN engine currently supports only single and double precision filters.");
    RuntimeError("Not implemented.");
}
template <>
typename CuDnnConvolutionEngineFactory<float>::FilterPtr CuDnnConvolutionEngineFactory<float>::CreateFilter(size_t w, size_t h, size_t c, size_t k)
{
    return std::make_unique<CuDnnFilter>(w, h, c, k, HIPDNN_DATA_FLOAT);
}
template <>
typename CuDnnConvolutionEngineFactory<double>::FilterPtr CuDnnConvolutionEngineFactory<double>::CreateFilter(size_t w, size_t h, size_t c, size_t k)
{
    return std::make_unique<CuDnnFilter>(w, h, c, k, HIPDNN_DATA_DOUBLE);
}

template <class ElemType>
typename CuDnnConvolutionEngineFactory<ElemType>::ConvDescPtr CuDnnConvolutionEngineFactory<ElemType>::CreateConvDescriptor(
    const Tensor4D& /*inT*/, const Filter& filterT, size_t wStride, size_t hStride, bool padding)
{
    size_t wPad = padding ? filterT.w() / 2 : 0;
    size_t hPad = padding ? filterT.h() / 2 : 0;
    return std::make_unique<CuDnnConvolutionDescriptor>(wStride, hStride, wPad, hPad);
}

template <class ElemType>
typename CuDnnConvolutionEngineFactory<ElemType>::PoolDescPtr CuDnnConvolutionEngineFactory<ElemType>::CreatePoolDescriptor(
    typename PoolDesc::PoolKind kind, size_t w, size_t h, size_t wStride, size_t hStride, size_t wPad, size_t hPad)
{
    return std::make_unique<CuDnnPoolingDescriptor>(kind, w, h, wStride, hStride, wPad, hPad);
}

template <class ElemType>
typename CuDnnConvolutionEngineFactory<ElemType>::ConvEnginePtr CuDnnConvolutionEngineFactory<ElemType>::CreateConvEngine(
    DEVICEID_TYPE deviceId, ImageLayoutKind imageLayout, size_t maxTempMemSizeInSamples, BatchNormImpl bnImpl)
{
    return std::make_unique<CuDnnConvolutionEngine<ElemType>>(deviceId, imageLayout, maxTempMemSizeInSamples, bnImpl);
}

template <class ElemType>
typename CuDnnConvolutionEngineFactory<ElemType>::PoolEnginePtr CuDnnConvolutionEngineFactory<ElemType>::CreatePoolEngine(
    DEVICEID_TYPE deviceId, ImageLayoutKind imageLayout)
{
    return std::make_unique<CuDnnPoolingEngine<ElemType>>(deviceId, imageLayout);
}

#else

template <class ElemType>
typename CuDnnConvolutionEngineFactory<ElemType>::Tensor4DPtr CuDnnConvolutionEngineFactory<ElemType>::CreateTensor(size_t, size_t, size_t, size_t)
{
    RuntimeError("The code is compiled without USE_CUDNN macro.");
}

template <class ElemType>
typename CuDnnConvolutionEngineFactory<ElemType>::FilterPtr CuDnnConvolutionEngineFactory<ElemType>::CreateFilter(size_t, size_t, size_t, size_t)
{
    RuntimeError("The code is compiled without USE_CUDNN macro.");
}

template <class ElemType>
typename CuDnnConvolutionEngineFactory<ElemType>::ConvDescPtr CuDnnConvolutionEngineFactory<ElemType>::CreateConvDescriptor(
    const Tensor4D&, const Filter&, size_t, size_t, bool)
{
    RuntimeError("The code is compiled without USE_CUDNN macro.");
}

template <class ElemType>
typename CuDnnConvolutionEngineFactory<ElemType>::PoolDescPtr CuDnnConvolutionEngineFactory<ElemType>::CreatePoolDescriptor(
    typename PoolDesc::PoolKind, size_t, size_t, size_t, size_t, size_t, size_t)
{
    RuntimeError("The code is compiled without USE_CUDNN macro.");
}

template <class ElemType>
typename CuDnnConvolutionEngineFactory<ElemType>::ConvEnginePtr CuDnnConvolutionEngineFactory<ElemType>::CreateConvEngine(DEVICEID_TYPE, ImageLayoutKind, size_t, BatchNormImpl)
{
    RuntimeError("The code is compiled without USE_CUDNN macro.");
}

template <class ElemType>
typename CuDnnConvolutionEngineFactory<ElemType>::PoolEnginePtr CuDnnConvolutionEngineFactory<ElemType>::CreatePoolEngine(DEVICEID_TYPE, ImageLayoutKind)
{
    RuntimeError("The code is compiled without USE_CUDNN macro.");
}

#endif

template class CuDnnConvolutionEngineFactory<float>;
template class CuDnnConvolutionEngineFactory<double>;
} } }
