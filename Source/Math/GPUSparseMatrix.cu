#include "hip/hip_runtime.h"
//
// Copyright (c) Microsoft. All rights reserved.
// Licensed under the MIT license. See LICENSE.md file in the project root for full license information.
//

#include "Basics.h"
#include "BestGpu.h"

#ifndef CPUONLY

#include "GPUSparseMatrix.h"
#include "GPUMatrix.h"
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include "hipblas.h"
#include "GPUMatrixCUDAKernels.cuh"
#include <functional>
#include "CommonMatrix.h"
#include <iostream> // for cout/cerr
#include <assert.h>

typedef unsigned char byte;

#pragma warning(disable : 4267) // conversion from 'size_t' to 'unsigned int'; happens in CUDA <<<a,b>>> syntax if a and b are size_t
#pragma warning(disable : 4127) // conditional expression is constant; "if (sizeof(ElemType)==sizeof(float))" triggers this

#ifdef _WIN32
// thread local storage to access the current stream, initalize to default stream
extern __declspec(thread)
#else
static
#endif
    hipStream_t t_stream;

template <>
const char* CudaErrString<hipsparseStatus_t>(hipsparseStatus_t)
{
    hipDeviceSynchronize();
    return "(see hipsparse.h & look for hipsparseStatus_t or CUSPARSE_STATUS_xxx)";
}

namespace Microsoft { namespace MSR { namespace CNTK {

#pragma region Constructors and Destructor

template <class ElemType>
GPUSPARSE_INDEX_TYPE GPUSparseMatrix<ElemType>::SecondaryIndexValueAt(size_t idx) const
{
    GPUSPARSE_INDEX_TYPE value;
    CUDA_CALL(hipMemcpy(&value, SecondaryIndexLocation() + idx, sizeof(GPUSPARSE_INDEX_TYPE), hipMemcpyDeviceToHost));

    return value;
}

//-------------------------------------------------------------------------
// construction and conversion
//-------------------------------------------------------------------------

template <class ElemType>
void GPUSparseMatrix<ElemType>::ZeroInit(const MatrixFormat matrixFormat, const DEVICEID_TYPE computeDevice)
{
    if (matrixFormat != MatrixFormat::matrixFormatSparseCSC && matrixFormat != MatrixFormat::matrixFormatSparseCSR &&
        matrixFormat != MatrixFormat::matrixFormatSparseBlockCol && matrixFormat != MatrixFormat::matrixFormatSparseBlockRow)
    {
        LogicError("GPUSparseMatrix:  unsupported sparse matrix format");
        // BUGBUG: Then why even define others?
    }
    Base::ZeroInit();
    m_computeDevice            = computeDevice; // current GPU device Id
    m_format                   = matrixFormat;

    m_sliceOf                  = nullptr;
    m_totalBufferSizeAllocated = 0;
    m_blockSize                = 0;
    m_rowToId                  = nullptr;
    m_tempHostBuffer           = nullptr;
    m_tempHostBufferSize       = 0;
}

template <class ElemType>
GPUSparseMatrix<ElemType>::GPUSparseMatrix(const size_t numRows, const size_t numCols, const size_t numNZ, DEVICEID_TYPE computeDevice, const MatrixFormat matrixFormat /*= MatrixFormat::matrixFormatSparseCSR*/)
{
    ZeroInit(matrixFormat, computeDevice);
    Resize(numRows, numCols, numNZ, true, false);
}

template <class ElemType>
GPUSparseMatrix<ElemType>::GPUSparseMatrix(DEVICEID_TYPE computeDevice, const MatrixFormat matrixFormat /*= MatrixFormat::matrixFormatSparseCSR*/)
{
    ZeroInit(matrixFormat, computeDevice);
}

template <class ElemType>
GPUSparseMatrix<ElemType>::GPUSparseMatrix(const GPUMatrix<ElemType>& deepCopy, const MatrixFormat matrixFormat /*= MatrixFormat::matrixFormatSparseCSR*/)
{
    ZeroInit(matrixFormat, deepCopy.GetComputeDeviceId());
    if (!deepCopy.IsEmpty())
        SetValue(deepCopy, matrixFormat);
}

template <class ElemType>
GPUSparseMatrix<ElemType>::GPUSparseMatrix(const GPUSparseMatrix<ElemType>& deepCopy)
{

    ZeroInit(deepCopy.GetFormat(), deepCopy.GetComputeDeviceId());
    DeepCopy(deepCopy);
}

// PrepareDevice - Setup the correct cuda context for an operation
// deviceId - the device on which the operation will take place
//            defaults to -1, which means use matrices current device
template <class ElemType>
DEVICEID_TYPE GPUSparseMatrix<ElemType>::PrepareDevice(DEVICEID_TYPE deviceId /*=-1*/) const
{
    // if default value use current compute device
    DEVICEID_TYPE newId = deviceId >= 0 ? deviceId : m_computeDevice;

    Microsoft::MSR::CNTK::PrepareDevice(newId);
    return newId;
}

template <class ElemType>
/*private*/ void GPUSparseMatrix<ElemType>::DeepCopy(const GPUSparseMatrix<ElemType>& deepCopy)
{
    ChangeDeviceTo(deepCopy.m_computeDevice);
    deepCopy.PrepareDevice();

    Resize(deepCopy.m_numRows, deepCopy.m_numCols, deepCopy.GetNumNZElements(), deepCopy.m_format, true, false);
    m_externalBuffer  = false;
    m_sliceOf         = nullptr;
    m_nz              = deepCopy.m_nz;
    m_sliceViewOffset = 0; // reset to zero as we only start copying the indices starting from the offset in the source matrix

    CUDA_CALL(hipMemcpy(BufferPointer(), deepCopy.NzValues(), NzSize(), hipMemcpyDeviceToDevice));
    CUDA_CALL(hipMemcpy(MajorIndexLocation(), deepCopy.MajorIndexLocationWithSliceViewOffset(), MajorIndexSize(), hipMemcpyDeviceToDevice));
    CUDA_CALL(hipMemcpy(SecondaryIndexLocation(), deepCopy.SecondaryIndexLocation(), SecondaryIndexSize(), hipMemcpyDeviceToDevice));

    if (deepCopy.m_sliceViewOffset > 0)
    {
        int blocksPerGrid = (int) ceil(1.0 * SecondaryIndexCount() / GridDim::maxThreadsPerBlock);
        SyncGuard syncGuard;
        _shiftColCSCIndexFromSliceViewToAbsolute<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(
            SecondaryIndexLocation(),
            SecondaryIndexCount(),
            GetNumNZElements());
    }

    // TODO: to copy other varibles used only for class based LM
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::SetValue(const GPUSparseMatrix<ElemType>& deepCopy)
{
    if (!OwnBuffer())
        LogicError("Cannot SetValue on managed external matrix");

    DeepCopy(deepCopy);
}

// from CPU
template <class ElemType>
void GPUSparseMatrix<ElemType>::SetValue(const CPUSparseMatrix<ElemType>& deepCopy)
{
    if (!OwnBuffer())
        LogicError("Cannot SetValue on Managed external matrix");

    SetFormat(deepCopy.GetFormat());
    if (deepCopy.IsEmpty())
    {
        Reset();
        return;
    }

    if (deepCopy.GetFormat() == matrixFormatSparseCSR)
    {
        SetMatrixFromCSRFormat(deepCopy.RowLocation(), deepCopy.ColLocation(), deepCopy.BufferPointer(), deepCopy.GetNumElemAllocated(), deepCopy.GetNumRows(), deepCopy.GetNumCols());
    }
    else if (deepCopy.GetFormat() == matrixFormatSparseCSC)
    {
        SetMatrixFromCSCFormat(deepCopy.ColLocation(), deepCopy.RowLocation(), deepCopy.BufferPointer(), deepCopy.GetNumElemAllocated(), deepCopy.GetNumRows(), deepCopy.GetNumCols());
    }
    else
        NOT_IMPLEMENTED;
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::CopyToCPUSparseMatrix(CPUSparseMatrix<ElemType>& cpuSparseMatrix) const
{
    if (!cpuSparseMatrix.OwnBuffer())
        LogicError("Cannot CopyToCPUSparseMatrix on managed external matrix");

    cpuSparseMatrix.SetFormat(GetFormat());
    if (IsEmpty())
    {
        cpuSparseMatrix.Reset();
        return;
    }

    if (this->GetFormat() == matrixFormatSparseCSR)
    {
        // we need to do conversion because CPUSparseMatrix uses size_t for indexes while GPUSparseMatrix uses int
        cpuSparseMatrix.Resize(GetNumRows(), GetNumCols(), GetNumElemAllocated(), true, false);
        cpuSparseMatrix.SetNzCount(GetNumNZElements());

        PrepareDevice();

        if (sizeof(GPUSPARSE_INDEX_TYPE) == sizeof(CPUSPARSE_INDEX_TYPE))
        {
            CUDA_CALL(hipMemcpy(cpuSparseMatrix.RowLocation(), RowLocation(), RowSize(), hipMemcpyDeviceToHost));
            CUDA_CALL(hipMemcpy(cpuSparseMatrix.ColLocation(), ColLocation(), ColSize(), hipMemcpyDeviceToHost));
        }
        else
        {
            GPUSPARSE_INDEX_TYPE* h_CSRRow = (GPUSPARSE_INDEX_TYPE*) ReserveTempHostBuffer(RowSize());
            CUDA_CALL(hipMemcpy(h_CSRRow, RowLocation(), RowSize(), hipMemcpyDeviceToHost));
            CopyBuffer(cpuSparseMatrix.RowLocation(), h_CSRRow, SecondaryIndexCount());

            GPUSPARSE_INDEX_TYPE* h_Col = (GPUSPARSE_INDEX_TYPE*) ReserveTempHostBuffer(ColSize());
            CUDA_CALL(hipMemcpy(h_Col, ColLocation(), ColSize(), hipMemcpyDeviceToHost));
            CopyBuffer(cpuSparseMatrix.ColLocation(), h_Col, MajorIndexCount());
        }

        CUDA_CALL(hipMemcpy(cpuSparseMatrix.BufferPointer(), BufferPointer(), GetSizeElemAllocated(), hipMemcpyDeviceToHost));
    }
    else if (this->GetFormat() == matrixFormatSparseCSC)
    {
        // we need to do conversion because CPUSparseMatrix uses size_t for indexes while GPUSparseMatrix uses int
        cpuSparseMatrix.Resize(GetNumRows(), GetNumCols(), GetNumNZElements(), true, false);
        cpuSparseMatrix.SetNzCount(GetNumNZElements());

        PrepareDevice();
        if (sizeof(GPUSPARSE_INDEX_TYPE) == sizeof(CPUSPARSE_INDEX_TYPE))
        {
            CUDA_CALL(hipMemcpy(cpuSparseMatrix.RowLocation(), RowLocation(), RowSize(), hipMemcpyDeviceToHost));
            CUDA_CALL(hipMemcpy(cpuSparseMatrix.ColLocation(), ColLocation(), ColSize(), hipMemcpyDeviceToHost));
        }
        else
        {
            GPUSPARSE_INDEX_TYPE* h_CSCCol = (GPUSPARSE_INDEX_TYPE*) ReserveTempHostBuffer(ColSize());
            CUDA_CALL(hipMemcpy(h_CSCCol, ColLocation(), ColSize(), hipMemcpyDeviceToHost));
            CopyBuffer(cpuSparseMatrix.ColLocation(), h_CSCCol, SecondaryIndexCount());

            GPUSPARSE_INDEX_TYPE* h_Row = (GPUSPARSE_INDEX_TYPE*) ReserveTempHostBuffer(RowSize());
            CUDA_CALL(hipMemcpy(h_Row, RowLocation(), RowSize(), hipMemcpyDeviceToHost));
            CopyBuffer(cpuSparseMatrix.RowLocation(), h_Row, MajorIndexCount());
        }

        CUDA_CALL(hipMemcpy(cpuSparseMatrix.BufferPointer(), BufferPointer(), GetSizeElemAllocated(), hipMemcpyDeviceToHost));
    }
    else
        NOT_IMPLEMENTED;
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::CopyToDenseMatrix(GPUMatrix<ElemType>& denseMatrix) const
{
    if (IsEmpty())
    {
        denseMatrix.Resize(0, 0);
        return;
    }

    PrepareDevice();
    hipsparseHandle_t cusparseHandle = 0;
    CUSPARSE_CALL(hipsparseCreate(&cusparseHandle));
    hipsparseMatDescr_t descr = 0;
    CUSPARSE_CALL(hipsparseCreateMatDescr(&descr));
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

    denseMatrix.Resize(m_numRows, m_numCols);

    SyncGuard syncGuard;
    CUSPARSE_CALL(hipsparseSetStream(cusparseHandle, t_stream));
    if (m_format == MatrixFormat::matrixFormatSparseCSR)
    {
        if (sizeof(ElemType) == sizeof(float))
        {
            CUSPARSE_CALL(hipsparseScsr2dense(cusparseHandle, int(m_numRows), int(m_numCols), descr, (float*) BufferPointer(), RowLocation(), ColLocation(), (float*) denseMatrix.BufferPointer(), int(m_numRows)));
        }
        else
        {
            CUSPARSE_CALL(hipsparseDcsr2dense(cusparseHandle, int(m_numRows), int(m_numCols), descr, (double*) BufferPointer(), RowLocation(), ColLocation(), (double*) denseMatrix.BufferPointer(), int(m_numRows)));
        }
    }
    else if (m_format == MatrixFormat::matrixFormatSparseCSC)
    {
        if (sizeof(ElemType) == sizeof(float))
        {
            CUSPARSE_CALL(hipsparseScsc2dense(cusparseHandle, int(m_numRows), int(m_numCols), descr, (float*) BufferPointer(), RowLocation(), ColLocation(), (float*) denseMatrix.BufferPointer(), int(m_numRows)));
        }
        else
        {
            CUSPARSE_CALL(hipsparseDcsc2dense(cusparseHandle, int(m_numRows), int(m_numCols), descr, (double*) BufferPointer(), RowLocation(), ColLocation(), (double*) denseMatrix.BufferPointer(), int(m_numRows)));
        }
    }
    else
    {
        NOT_IMPLEMENTED;
    }
    CUSPARSE_CALL(hipsparseDestroy(cusparseHandle));

}

template <class ElemType>
void GPUSparseMatrix<ElemType>::ConvertToSparseFormat(MatrixFormat newFormat, GPUSparseMatrix<ElemType>& outMatrix) const
{
    if (!outMatrix.OwnBuffer())
        LogicError("Cannot ConvertToSparseFormat to managed external matrix");

    if (IsEmpty())
    {
        outMatrix.ZeroInit(newFormat, GetComputeDeviceId());
        return;
    }

    MatrixFormat oldFormat = GetFormat();
    if (oldFormat == newFormat)
    {
        outMatrix.SetValue(*this);
        return;
    }

    PrepareDevice();
    hipsparseHandle_t cusparseHandle = 0;
    CUSPARSE_CALL(hipsparseCreate(&cusparseHandle));

    SyncGuard syncGuard;
    CUSPARSE_CALL(hipsparseSetStream(cusparseHandle, t_stream));

    outMatrix.ChangeDeviceTo(GetComputeDeviceId());
    outMatrix.Resize(m_numRows, m_numCols, m_nz, newFormat, true, false);
    outMatrix.SetNzCount(m_nz);

    if ((oldFormat == matrixFormatSparseCSR && newFormat == matrixFormatSparseCSC) || (oldFormat == matrixFormatSparseCSC && newFormat == matrixFormatSparseCSR))
    {
        if (sizeof(ElemType) == sizeof(float))
        {
            CUSPARSE_CALL(hipsparseScsr2csc(cusparseHandle, int(m_numRows), int(m_numCols), int(m_elemSizeAllocated),
                                           (float*) BufferPointer(), RowLocation(), ColLocation(), (float*) outMatrix.BufferPointer(),
                                           outMatrix.RowLocation(), outMatrix.ColLocation(), HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO));
        }
        else
        {
            CUSPARSE_CALL(hipsparseDcsr2csc(cusparseHandle, int(m_numRows), int(m_numCols), int(m_elemSizeAllocated),
                                           (double*) BufferPointer(), RowLocation(), ColLocation(), (double*) outMatrix.BufferPointer(),
                                           outMatrix.RowLocation(), outMatrix.ColLocation(), HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO));
        }
    }
    else
    {
        NOT_IMPLEMENTED;
    }

    CUSPARSE_CALL(hipsparseDestroy(cusparseHandle));
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::ConvertToSparseFormat(MatrixFormat newFormat)
{
    if (IsEmpty())
    {
        SetFormat(newFormat);
        return;
    }

    MatrixFormat oldFormat = GetFormat();
    if (oldFormat == newFormat)
        return;

    GPUSparseMatrix<ElemType> tempMatrix(GetComputeDeviceId(), newFormat);
    ConvertToSparseFormat(newFormat, tempMatrix);

    *this = std::move(tempMatrix);
}

template <class ElemType>
GPUMatrix<ElemType> GPUSparseMatrix<ElemType>::CopyToDenseMatrix() const
{
    GPUMatrix<ElemType> res(GetComputeDeviceId());
    if (!IsEmpty())
        CopyToDenseMatrix(res);
    return res;
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::ChangeDeviceTo(DEVICEID_TYPE to_id)
{
    if (!OwnBuffer())
        LogicError("Cannot change device on Managed external matrix");
    if (to_id == CPUDEVICE)
        LogicError("to_id must be valid GPU");
    if (m_computeDevice == to_id)
        return;

    if (m_totalBufferSizeAllocated == 0) // nothing to move
    {
        assert(m_pArray == nullptr);
    }
    else
    {
        ElemType* d_dst = reinterpret_cast<ElemType*>(TracingGPUMemoryAllocator::Allocate<char>(to_id, m_totalBufferSizeAllocated));

        // first try peer access
        int canAccessPeer = false;
        CUDA_CALL(hipDeviceCanAccessPeer(&canAccessPeer, to_id, m_computeDevice));
        if (canAccessPeer)
        {
            hipError_t cudaStatus = hipDeviceEnablePeerAccess(m_computeDevice, 0);
            if (cudaStatus != hipErrorPeerAccessAlreadyEnabled)
            {
                CUDA_CALL(cudaStatus);
            }
            CUDA_CALL(hipMemcpyPeer(d_dst, to_id, m_pArray, m_computeDevice, m_totalBufferSizeAllocated));
        }
        else
        {
            // peer access didn't work, just copy normal
            // make this more efficient by keeping some buffers available for each copy
            ElemType* h_dst = NULL;
            PrepareDevice();
            CUDA_CALL(hipHostMalloc((void**) &h_dst, m_totalBufferSizeAllocated));
            CUDA_CALL(hipMemcpy(h_dst, m_pArray, m_totalBufferSizeAllocated, hipMemcpyDeviceToHost));
            PrepareDevice((DEVICEID_TYPE) to_id);
            CUDA_CALL(hipMemcpy(d_dst, h_dst, m_totalBufferSizeAllocated, hipMemcpyHostToDevice));
            CUDA_CALL(hipHostFree(h_dst));
        }

        TracingGPUMemoryAllocator::Free<ElemType>(m_computeDevice, m_pArray);
        m_pArray = d_dst;
    }

    SetComputeDeviceId(PrepareDevice(to_id));
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::SetValue(const GPUMatrix<ElemType>& denseMatrix)
{
    if (!OwnBuffer())
        LogicError("Cannot SetValue on Managed external matrix");

    SetValue(denseMatrix, GetFormat());
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::SetValue(const GPUMatrix<ElemType>& denseMatrix, const MatrixFormat matrixFormat)
{
    if (!OwnBuffer())
        LogicError("Cannot SetValue on Managed external matrix");

    if (matrixFormat != matrixFormatSparseCSR && matrixFormat != matrixFormatSparseCSC)
    {
        NOT_IMPLEMENTED;
    }

    PrepareDevice();
    hipsparseHandle_t cusparseHandle = 0;
    CUSPARSE_CALL(hipsparseCreate(&cusparseHandle));
    hipsparseMatDescr_t descr = 0;
    CUSPARSE_CALL(hipsparseCreateMatDescr(&descr));
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

    int numRows = (int) denseMatrix.GetNumRows(); // m
    int numCols = (int) denseMatrix.GetNumCols(); // n

    int* nnzPerRowOrCol = TracingGPUMemoryAllocator::Allocate<GPUSPARSE_INDEX_TYPE>(m_computeDevice, ((matrixFormat & matrixFormatRowMajor) ? numRows : numCols));
    int nnzTotalDevHostPtr = -1;

    {
        SyncGuard syncGuard;
        if (sizeof(ElemType) == sizeof(float))
        {
            CUSPARSE_CALL(hipsparseSnnz(cusparseHandle, (matrixFormat & matrixFormatRowMajor) ? HIPSPARSE_DIRECTION_ROW : HIPSPARSE_DIRECTION_COLUMN, (int) numRows, (int) numCols, descr,
                                       reinterpret_cast<float*>(denseMatrix.BufferPointer()), (int) numRows, nnzPerRowOrCol, &nnzTotalDevHostPtr));
        }
        else
        {
            CUSPARSE_CALL(hipsparseDnnz(cusparseHandle, (matrixFormat & matrixFormatRowMajor) ? HIPSPARSE_DIRECTION_ROW : HIPSPARSE_DIRECTION_COLUMN, (int) numRows, (int) numCols, descr,
                                       reinterpret_cast<double*>(denseMatrix.BufferPointer()), (int) numRows, nnzPerRowOrCol, &nnzTotalDevHostPtr));
        }
        // ~SyncGuard
    }

    Resize(numRows, numCols, nnzTotalDevHostPtr, matrixFormat, true, false);
    SetNzCount(nnzTotalDevHostPtr);

    SyncGuard syncGuard;
    if (m_format == MatrixFormat::matrixFormatSparseCSR)
    {
        if (sizeof(ElemType) == sizeof(float))
        {
            CUSPARSE_CALL(hipsparseSdense2csr(cusparseHandle, (int) m_numRows, (int) m_numCols, descr, reinterpret_cast<float*>(denseMatrix.BufferPointer()),
                                             (int) m_numRows, nnzPerRowOrCol, reinterpret_cast<float*>(BufferPointer()), RowLocation(), ColLocation()));
        }
        else
        {
            CUSPARSE_CALL(hipsparseDdense2csr(cusparseHandle, (int) m_numRows, (int) m_numCols, descr, reinterpret_cast<double*>(denseMatrix.BufferPointer()),
                                             (int) m_numRows, nnzPerRowOrCol, reinterpret_cast<double*>(BufferPointer()), RowLocation(), ColLocation()));
        }
    }
    else if (m_format == MatrixFormat::matrixFormatSparseCSC)
    {
        if (sizeof(ElemType) == sizeof(float))
        {
            CUSPARSE_CALL(hipsparseSdense2csc(cusparseHandle, (int) m_numRows, (int) m_numCols, descr, reinterpret_cast<float*>(denseMatrix.BufferPointer()),
                                             (int) m_numRows, nnzPerRowOrCol, reinterpret_cast<float*>(BufferPointer()), RowLocation(), ColLocation()));
        }
        else
        {
            CUSPARSE_CALL(hipsparseDdense2csc(cusparseHandle, (int) m_numRows, (int) m_numCols, descr, reinterpret_cast<double*>(denseMatrix.BufferPointer()),
                                             (int) m_numRows, nnzPerRowOrCol, reinterpret_cast<double*>(BufferPointer()), RowLocation(), ColLocation()));
        }
    }
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::operator=(const GPUSparseMatrix<ElemType>& deepCopy)
{
    Clear();
    if (this != &deepCopy)
        SetValue(deepCopy);

    return *this;
}

template <class ElemType>
GPUSparseMatrix<ElemType>::GPUSparseMatrix(GPUSparseMatrix<ElemType>&& moveFrom)
{
    Base::ShallowCopyFrom(moveFrom);
    // TODO: implement this using operator= or a shared function
    m_totalBufferSizeAllocated = moveFrom.m_totalBufferSizeAllocated;
    m_sliceOf                  = moveFrom.m_sliceOf;
    m_blockSize                = moveFrom.m_blockSize;
    m_rowToId                  = moveFrom.m_rowToId;
    m_tempHostBuffer           = moveFrom.m_tempHostBuffer;
    m_tempHostBufferSize       = moveFrom.m_tempHostBufferSize;
    moveFrom.ZeroInit(moveFrom.m_format, moveFrom.m_computeDevice); // so that memory in moveFrom is not freed
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::operator=(GPUSparseMatrix<ElemType>&& moveFrom)
{
    if (this != &moveFrom)
    {
        if (OwnBuffer())
            ReleaseMemory(); // always delete the data pointer since we will use the pointer from moveFrom
        Base::ShallowCopyFrom(moveFrom);
        m_totalBufferSizeAllocated = moveFrom.m_totalBufferSizeAllocated;
        m_sliceOf                  = moveFrom.m_sliceOf;
        m_blockSize                = moveFrom.m_blockSize;
        m_rowToId                  = moveFrom.m_rowToId;
        m_tempHostBuffer           = moveFrom.m_tempHostBuffer;
        m_tempHostBufferSize       = moveFrom.m_tempHostBufferSize;
        moveFrom.ZeroInit(moveFrom.m_format, moveFrom.m_computeDevice);
    }

    return *this;
}

template <class ElemType>
GPUSparseMatrix<ElemType>::~GPUSparseMatrix()
{
    ReleaseMemory();
}

template <class ElemType>
/*private*/ void GPUSparseMatrix<ElemType>::ReleaseMemory()
{
    // If OwnBuffer() is false then this matrix is a view over another matrix.
    // In that case we shouldn't free anything.
    if (OwnBuffer())
    {
        delete[](byte*) m_tempHostBuffer;
        m_tempHostBuffer = nullptr;

        TracingGPUMemoryAllocator::Free<ElemType>(m_computeDevice, m_pArray);
        m_pArray = nullptr;

        TracingGPUMemoryAllocator::Free<GPUSPARSE_INDEX_TYPE>(m_computeDevice, m_rowToId);
        m_rowToId = nullptr;
    }
    ZeroInit(m_format, m_computeDevice);
}

//ResizeAsAndCopyIndexFrom - Resize this sparse matrix to have the same element structure as the passed matrix
// a - sparse matrix whose structure we want to clone
// remark: this was done for element wise operations where the structure will be identical after an operation
template <class ElemType>
void GPUSparseMatrix<ElemType>::ResizeAsAndCopyIndexFrom(const GPUSparseMatrix<ElemType>& a, const bool growOnly /*= true*/)
{
    Resize(a.m_numRows, a.m_numCols, a.m_nz, a.m_format, growOnly, false);
    SetNzCount(a.m_nz);

    CUDA_CALL(hipMemcpy(MajorIndexLocation(), a.MajorIndexLocation(), MajorIndexSize(), hipMemcpyDeviceToDevice));
    CUDA_CALL(hipMemcpy(SecondaryIndexLocation(), a.SecondaryIndexLocation(), SecondaryIndexSize(), hipMemcpyDeviceToDevice));
}

//-------------------------------------------------------------------------
// main operations
//-------------------------------------------------------------------------

template <class ElemType>
void GPUSparseMatrix<ElemType>::Reshape(const size_t numRows, const size_t numCols)
{
    if (m_numRows == numRows && m_numCols == numCols)
        return;

    if (!OwnBuffer())
        LogicError("GPUSparseMatrix::Reshape: Cannot Reshape since the buffer is managed externally.");

    if (m_format != MatrixFormat::matrixFormatSparseCSC)
        NOT_IMPLEMENTED;

    if (m_numRows * m_numCols != numRows * numCols)
        LogicError("GPUSparseMatrix::Reshape: new matrix size does not match current size, can't be reshaped. Did you mean to resize?");

    size_t bufferSizeNeeded = BufferSizeNeeded(numRows, numCols, m_elemSizeAllocated, m_format);

    ElemType* pArray = reinterpret_cast<ElemType*>(TracingGPUMemoryAllocator::Allocate<char>(m_computeDevice, bufferSizeNeeded));

    if (m_pArray != nullptr)
    {
        CUDA_CALL(hipMemcpy(pArray, BufferPointer(), GetSizeElemAllocated(), hipMemcpyDeviceToDevice));

        GPUSPARSE_INDEX_TYPE* majorIndexInNewBuffer = (GPUSPARSE_INDEX_TYPE*) (pArray + m_elemSizeAllocated);
        GPUSPARSE_INDEX_TYPE* secondaryIndexInNewBuffer = majorIndexInNewBuffer + MajorIndexCount(numRows, numCols, m_elemSizeAllocated, m_format);

        int blocksPerGrid = (int) ceil(1.0 * numCols / GridDim::maxThreadsPerBlock);
        SyncGuard syncGuard;
        _reshape<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(
            m_numRows,                // old row count
            m_numCols,                // old col count
            numRows,                  // new row count
            numCols,                  // new col count
            MajorIndexLocation(),     // old row index array
            SecondaryIndexLocation(), // old column index array
            majorIndexInNewBuffer,    // new row index array
            secondaryIndexInNewBuffer // new column index array
            );
        TracingGPUMemoryAllocator::Free<ElemType>(m_computeDevice, m_pArray);
    }

    m_pArray = pArray;
    m_numRows = numRows;
    m_numCols = numCols;
    m_totalBufferSizeAllocated = bufferSizeNeeded;

    // following are generated dynamically and no need to save
    if (m_rowToId != nullptr)
        TracingGPUMemoryAllocator::Free<GPUSPARSE_INDEX_TYPE>(m_computeDevice, m_rowToId);

    m_rowToId = TracingGPUMemoryAllocator::Allocate<GPUSPARSE_INDEX_TYPE>(m_computeDevice, m_elemSizeAllocated);
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::Resize(const size_t numRows, const size_t numCols, const size_t numNZElemToReserve, const bool growOnly, bool keepExistingValues)
{
    Resize(numRows, numCols, numNZElemToReserve, GetFormat(), growOnly, keepExistingValues);
}

// WARNING: When memory is reallocated, existing information will be lost.
// TODO: add keepExistingValues (default to true) argument so that the existing values are kept even after reallocation
template <class ElemType>
void GPUSparseMatrix<ElemType>::Resize(const size_t numRows, const size_t numCols, const size_t numNZElemToReserve, const MatrixFormat matrixFormat, const bool growOnly /*= true*/, bool keepExistingValues /*=true*/)
{
    if (!OwnBuffer())
        LogicError("Cannot Resize since the buffer is managed externally.");

    if (matrixFormat != m_format || m_numRows != numRows || m_numCols != numCols)
        keepExistingValues = false;

    size_t bufferSizeNeeded = BufferSizeNeeded(numRows, numCols, numNZElemToReserve, matrixFormat);
    bool reallocate = (m_totalBufferSizeAllocated < bufferSizeNeeded || (!growOnly && m_totalBufferSizeAllocated > bufferSizeNeeded));

    if (reallocate)
    {
        ElemType* pArray = reinterpret_cast<ElemType*>(TracingGPUMemoryAllocator::Allocate<char>(m_computeDevice, bufferSizeNeeded));

        if (m_pArray != nullptr)
        {
            if (keepExistingValues)
            {
                if (m_nz > numNZElemToReserve || m_totalBufferSizeAllocated > bufferSizeNeeded)
                    LogicError("Resize: To keep values m_nz should <= numNZElemToReserve.");

                CUDA_CALL(hipMemcpy(pArray, BufferPointer(), GetSizeElemAllocated(), hipMemcpyDeviceToDevice));

                GPUSPARSE_INDEX_TYPE* majorIndexInNewBuffer = (GPUSPARSE_INDEX_TYPE*) (pArray + numNZElemToReserve);

                CUDA_CALL(hipMemcpy(majorIndexInNewBuffer, MajorIndexLocation(), MajorIndexSize(), hipMemcpyDeviceToDevice));

                GPUSPARSE_INDEX_TYPE* secondaryIndexInNewBuffer = majorIndexInNewBuffer + MajorIndexCount(numRows, numCols, numNZElemToReserve, matrixFormat);
                CUDA_CALL(hipMemcpy(secondaryIndexInNewBuffer, SecondaryIndexLocation(), SecondaryIndexSize(), hipMemcpyDeviceToDevice));
            }
            else
                m_nz = 0;

            TracingGPUMemoryAllocator::Free<ElemType>(m_computeDevice, m_pArray);
        }

        m_pArray = pArray;

        // following are generated dynamically and no need to save
        if (m_rowToId != nullptr)
            TracingGPUMemoryAllocator::Free<GPUSPARSE_INDEX_TYPE>(m_computeDevice, m_rowToId);

        m_rowToId = TracingGPUMemoryAllocator::Allocate<GPUSPARSE_INDEX_TYPE>(m_computeDevice, numNZElemToReserve);

        m_totalBufferSizeAllocated = bufferSizeNeeded;
        m_elemSizeAllocated = numNZElemToReserve;
    }
    else // if requested size is smaller, keeping original values does not make sense
    {
        m_elemSizeAllocated = ElemCountFromBufferSize(numRows, numCols, matrixFormat, m_totalBufferSizeAllocated);
    }

    m_numRows = numRows;
    m_numCols = numCols;
    m_format = matrixFormat;
}

// Reset matrix to 0.
template <class ElemType>
void GPUSparseMatrix<ElemType>::Reset()
{
    if (!OwnBuffer())
        LogicError("Cannot Reset since the buffer is managed externally.");

    m_nz = 0;
    m_blockSize = 0;
}

// copy features to GPU
template <class ElemType>
void GPUSparseMatrix<ElemType>::SetMatrixFromCSRFormat(const GPUSPARSE_INDEX_TYPE* h_CSRRow, const GPUSPARSE_INDEX_TYPE* h_Col, const ElemType* h_Val,
                                                       const size_t nz, const size_t numRows, const size_t numCols, const bool IsOnDevice /*= false*/, const DEVICEID_TYPE devId /*= -1*/)
{
    if (!OwnBuffer())
        LogicError("Cannot Set since the buffer is managed externally.");

    if (h_CSRRow == nullptr || h_Col == nullptr || h_Val == nullptr)
        LogicError("SetMatrixFromCSRFormat: nullptr passed in.");

    SetComputeDeviceId(PrepareDevice(devId));

    m_format = matrixFormatSparseCSR;
    Resize(numRows, numCols, nz, true, false);
    SetNzCount(nz);

    hipMemcpyKind kind = IsOnDevice ? hipMemcpyDeviceToDevice : hipMemcpyHostToDevice;
    CUDA_CALL(hipMemcpy(BufferPointer(), h_Val, NzSize(), kind));

    if (sizeof(CPUSPARSE_INDEX_TYPE) == sizeof(GPUSPARSE_INDEX_TYPE))
    {
        CUDA_CALL(hipMemcpy(RowLocation(), h_CSRRow, RowSize(), kind));
        CUDA_CALL(hipMemcpy(ColLocation(), h_Col, ColSize(), kind));
    }
    else
    {
        GPUSPARSE_INDEX_TYPE* pCol = (GPUSPARSE_INDEX_TYPE*) ReserveTempHostBuffer(RowSize() + ColSize());
        CopyBuffer(pCol, h_Col, MajorIndexCount());

        GPUSPARSE_INDEX_TYPE* pRow = pCol + MajorIndexCount();
        CopyBuffer(pRow, h_CSRRow, SecondaryIndexCount());

        CUDA_CALL(hipMemcpy(RowLocation(), pRow, RowSize(), kind));
        CUDA_CALL(hipMemcpy(ColLocation(), pCol, ColSize(), kind));
    }
}

// this function will allocate memory while the caller needs to release it
template <class ElemType>
void GPUSparseMatrix<ElemType>::GetMatrixFromCSRFormat(CPUSPARSE_INDEX_TYPE*& h_CSRRow, CPUSPARSE_INDEX_TYPE*& h_Col, ElemType*& h_Val, size_t& numElemAllocated, size_t& nz, size_t& numRows, size_t& numCols) const
{
    if (!OwnBuffer())
        LogicError("Cannot Set since the buffer is managed externally.");

    if (h_CSRRow != nullptr || h_Col != nullptr || h_Val != nullptr)
        LogicError("GetMatrixFromCSRFormat: Passed pointers must be nullptr");

    numElemAllocated = GetNumElemAllocated();
    nz = GetNumNZElements();
    numRows = GetNumRows();
    numCols = GetNumCols();

    if (IsEmpty() || nz == 0)
        return;
    else
    {
        h_Val = new ElemType[numElemAllocated];
        h_CSRRow = new CPUSPARSE_INDEX_TYPE[m_numRows + 1];
        h_Col = new CPUSPARSE_INDEX_TYPE[nz];

        PrepareDevice();
        CUDA_CALL(hipMemcpy(h_Val, BufferPointer(), GetSizeElemAllocated(), hipMemcpyDeviceToHost));

        if (sizeof(CPUSPARSE_INDEX_TYPE) == sizeof(GPUSPARSE_INDEX_TYPE))
        {
            CUDA_CALL(hipMemcpy(h_CSRRow, RowLocation(), RowSize(), hipMemcpyDeviceToHost));
            CUDA_CALL(hipMemcpy(h_Col, ColLocation(), ColSize(), hipMemcpyDeviceToHost));
        }
        else
        {
            GPUSPARSE_INDEX_TYPE* pCol = (GPUSPARSE_INDEX_TYPE*) ReserveTempHostBuffer(RowSize() + ColSize());
            GPUSPARSE_INDEX_TYPE* pRow = pCol + MajorIndexCount();

            CUDA_CALL(hipMemcpy(pRow, RowLocation(), RowSize(), hipMemcpyDeviceToHost));
            CUDA_CALL(hipMemcpy(pCol, ColLocation(), ColSize(), hipMemcpyDeviceToHost));

            CopyBuffer(h_Col, pCol, MajorIndexCount());
            CopyBuffer(h_CSRRow, pRow, SecondaryIndexCount());
        }
    }
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::SetMatrixFromCSCFormat(const CPUSPARSE_INDEX_TYPE* h_CSCCol, const CPUSPARSE_INDEX_TYPE* h_Row, const ElemType* h_Val,
                                                       const size_t nz, const size_t numRows, const size_t numCols, const bool IsOnDevice /*= false*/, const DEVICEID_TYPE devId /*= -1*/)
{
    if (!OwnBuffer())
        LogicError("Cannot Set since the buffer is managed externally.");

    if (h_CSCCol == nullptr || h_Row == nullptr || h_Val == nullptr)
        LogicError("SetMatrixFromCSCFormat: nullptr passed in.");

    SetComputeDeviceId(PrepareDevice(devId));
    m_format = matrixFormatSparseCSC;
    Resize(numRows, numCols, nz, true, false);
    SetNzCount(nz);

    hipMemcpyKind kind = IsOnDevice ? hipMemcpyDeviceToDevice : hipMemcpyHostToDevice;
    CUDA_CALL(hipMemcpy(BufferPointer(), h_Val, NzSize(), kind));

    if (sizeof(CPUSPARSE_INDEX_TYPE) == sizeof(GPUSPARSE_INDEX_TYPE))
    {
        CUDA_CALL(hipMemcpy(RowLocation(), h_Row, RowSize(), kind));
        CUDA_CALL(hipMemcpy(ColLocation(), h_CSCCol, ColSize(), kind));
    }
    else
    {
        GPUSPARSE_INDEX_TYPE* pCol = (GPUSPARSE_INDEX_TYPE*) ReserveTempHostBuffer(RowSize() + ColSize());
        GPUSPARSE_INDEX_TYPE* pRow = pCol + SecondaryIndexCount();

        CopyBuffer(pCol, h_CSCCol, SecondaryIndexCount());
        CopyBuffer(pRow, h_Row, MajorIndexCount());

        CUDA_CALL(hipMemcpy(RowLocation(), pRow, RowSize(), kind));
        CUDA_CALL(hipMemcpy(ColLocation(), pCol, ColSize(), kind));
    }
}

// this function will allocate memory while the caller needs to release it
template <class ElemType>
void GPUSparseMatrix<ElemType>::GetMatrixFromCSCFormat(GPUSPARSE_INDEX_TYPE*& h_CSCCol, GPUSPARSE_INDEX_TYPE*& h_Row, ElemType*& h_Val, size_t& numElemAllocated, size_t& nz, size_t& numRows, size_t& numCols) const
{
    if (h_CSCCol != nullptr || h_Row != nullptr || h_Val != nullptr)
        LogicError("GetMatrixFromCSCFormat: Passed pointers must be nullptr");

    numElemAllocated = GetNumElemAllocated();
    nz = GetNumNZElements();
    numRows = GetNumRows();
    numCols = GetNumCols();

    if (IsEmpty())
        return;
    else
    {
        h_Val = new ElemType[numElemAllocated];
        h_CSCCol = new GPUSPARSE_INDEX_TYPE[m_numRows + 1];
        h_Row = new GPUSPARSE_INDEX_TYPE[nz];

        PrepareDevice();
        CUDA_CALL(hipMemcpy(h_Val, BufferPointer(), GetSizeElemAllocated(), hipMemcpyDeviceToHost));

        if (sizeof(CPUSPARSE_INDEX_TYPE) == sizeof(GPUSPARSE_INDEX_TYPE))
        {
            CUDA_CALL(hipMemcpy(h_Row, RowLocation(), RowSize(), hipMemcpyDeviceToHost));
            CUDA_CALL(hipMemcpy(h_CSCCol, ColLocation(), ColSize(), hipMemcpyDeviceToHost));
        }
        else
        {
            GPUSPARSE_INDEX_TYPE* pCol = (GPUSPARSE_INDEX_TYPE*) ReserveTempHostBuffer(RowSize() + ColSize());
            GPUSPARSE_INDEX_TYPE* pRow = pCol + SecondaryIndexCount();

            CUDA_CALL(hipMemcpy(pRow, RowLocation(), RowSize(), hipMemcpyDeviceToHost));
            CUDA_CALL(hipMemcpy(pCol, ColLocation(), ColSize(), hipMemcpyDeviceToHost));

            CopyBuffer(h_CSCCol, pCol, SecondaryIndexCount());
            CopyBuffer(h_Row, pRow, MajorIndexCount());
        }
    }
}

#pragma endregion Constructors and Destructor

#pragma region Static BLAS Functions

// dense X sparse = dense
template <class ElemType>
void GPUSparseMatrix<ElemType>::MultiplyAndWeightedAdd(ElemType alpha, const GPUMatrix<ElemType>& lhs, const bool transposeA,
                                                       const GPUSparseMatrix<ElemType>& rhs, const bool transposeB, ElemType beta, GPUMatrix<ElemType>& c)
{
    if (lhs.GetComputeDeviceId() != rhs.GetComputeDeviceId() || (lhs.GetComputeDeviceId() != c.GetComputeDeviceId()))
        RuntimeError("GPUSparseMatrix::MultiplyAndWeightedAdd: All matrices must be on the same GPU");

    if (lhs.IsEmpty() || rhs.IsEmpty())
        LogicError("GPUSparseMatrix::MultiplyAndWeightedAdd:  one of the input matrix is empty.");

    int m = transposeA ? (int) lhs.GetNumCols() : (int) lhs.GetNumRows();
    int k = transposeA ? (int) lhs.GetNumRows() : (int) lhs.GetNumCols();
    int l = transposeB ? (int) rhs.GetNumCols() : (int) rhs.GetNumRows();
    int n = transposeB ? (int) rhs.GetNumRows() : (int) rhs.GetNumCols();

    assert(m > 0 && k > 0 && l > 0 && n > 0); // converting from size_t to int may cause overflow
    assert(k == l);
    if (k != l)
    {
        InvalidArgument("GPUSparseMatrix::MultiplyAndWeightedAdd: The inner dimensions of a and b must match.");
    }

    if (beta == 0)
        c.Resize(m, n);
    else
        c.VerifySize(m, n); // Can't resize if beta != 0

    c.PrepareDevice();
    if (rhs.m_format == MatrixFormat::matrixFormatSparseCSC)
    {
        ConvolveAndWeightedAdd(alpha, lhs, transposeA, rhs, transposeB, beta, c, 1, 1, false, false);
    }
    else if (rhs.m_format == matrixFormatSparseCSR)
    {
        GPUSparseMatrix<ElemType> tempMatrix(rhs.GetComputeDeviceId(), matrixFormatSparseCSC);
        rhs.ConvertToSparseFormat(matrixFormatSparseCSC, tempMatrix);
        MultiplyAndWeightedAdd(alpha, lhs, transposeA, tempMatrix, transposeB, beta, c);
    }
    else
    {
        NOT_IMPLEMENTED;
    }
}

// dense X sparse = dense
template <class ElemType>
void GPUSparseMatrix<ElemType>::ConvolveAndWeightedAdd(ElemType alpha, const GPUMatrix<ElemType>& lhs, const bool transposeA,
                                                       const GPUSparseMatrix<ElemType>& rhs, const bool transposeB, ElemType beta,
                                                       GPUMatrix<ElemType>& c, size_t numChannels, size_t horizontalSubsample, bool padding, bool channelwise)
{
    if (lhs.GetComputeDeviceId() != rhs.GetComputeDeviceId() || (lhs.GetComputeDeviceId() != c.GetComputeDeviceId()))
        RuntimeError("GPUSparseMatrix<ElemType>::ConvolveAndWeightedAdd: All matrices must be on the same GPU");

    if (lhs.IsEmpty() || rhs.IsEmpty())
        LogicError("GPUSparseMatrix<ElemType>::ConvolveAndWeightedAdd:  one of the input matrix is empty.");

    int m = transposeA ? (int) lhs.GetNumCols() : (int) lhs.GetNumRows();
    int k = transposeA ? (int) lhs.GetNumRows() : (int) lhs.GetNumCols();
    int l = transposeB ? (int) rhs.GetNumCols() : (int) rhs.GetNumRows();
    int n = transposeB ? (int) rhs.GetNumRows() : (int) rhs.GetNumCols();

    assert(m > 0 && k > 0 && l > 0 && n > 0); // converting from size_t to int may cause overflow

    int numSteps = 0;
    if (padding)
        numSteps = (int) ceil(1.0 * l / (horizontalSubsample * numChannels));
    else if (l >= k)
        numSteps = 1 + (l - k) / (horizontalSubsample * numChannels);

    if (numSteps == 0)
        LogicError("ConvolveAndWeightedAdd: number of steps is zero. Matrix dimensions are incorrect or set padding to true.");

    int cRows = m * numSteps;
    int cCols = n;

    if (beta == 0)
        c.Resize(cRows, cCols);
    else
        c.VerifySize(cRows, cCols); // Can't resize if beta != 0

    c.PrepareDevice();
    if (rhs.m_format == MatrixFormat::matrixFormatSparseCSC)
    {
        if (!transposeB)
        {
            int blocksPerGrid = (int) ceil(1.0 * cRows * cCols / GridDim::maxThreadsPerBlock);
            SyncGuard syncGuard;
            _dense1DConvMultSparseCSCAndWeightedAddToDense<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(
                m,                   // rowDense
                k,                   // colDense
                n,                   // colSparse
                numChannels,         // number of input channels
                numSteps,            // convolution num steps
                horizontalSubsample, // convolution step size
                channelwise,         // channelwise or pixelwise multiplication
                alpha,
                reinterpret_cast<const ElemType*>(lhs.BufferPointer()), // dense
                transposeA,
                reinterpret_cast<const ElemType*>(rhs.BufferPointer()), // sparse nz values
                rhs.RowLocation(),
                rhs.ColLocation(),
                beta,
                reinterpret_cast<ElemType*>(c.BufferPointer()) // dense target
                );
        }
        else
        {
            if (beta != 1.0)
            {
                RuntimeError("Only support c += alpha * a operation");
            }
            int blocksPerGrid = (int) ceil(1.0 * cRows / GridDim::maxThreadsPerBlock);
            SyncGuard syncGuard;
            for (int rowInB = 0; rowInB < l; rowInB++)
            {
                _dense1DConvMultSparseCSCTransposeAndAddToDense<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(
                    m,                   // rowDense
                    k,                   // colDense
                    n,                   // colSparse
                    numChannels,         // number of input channels
                    numSteps,            // convolution num steps
                    horizontalSubsample, // convolution step size
                    channelwise,         // channelwise or pixelwise multiplication
                    rowInB,
                    alpha,
                    reinterpret_cast<const ElemType*>(lhs.BufferPointer()), // dense
                    transposeA,
                    reinterpret_cast<const ElemType*>(rhs.BufferPointer()), // sparse nz values
                    rhs.RowLocation(),
                    rhs.ColLocation(),
                    reinterpret_cast<ElemType*>(c.BufferPointer()) // dense target
                    );
            }
        }
    }
    else
    {
        NOT_IMPLEMENTED;
    }
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::TensorShuffleScaleAndAdd(ElemType keepWeight, const GPUSparseMatrix<ElemType>& a, size_t D, size_t S, size_t M, size_t K, size_t T, ElemType scaleFactor, const GPUSparseMatrix<ElemType>& b, GPUSparseMatrix<ElemType>& c)
{
    if (!c.OwnBuffer())
        LogicError("Cannot modify externally managed matrix");

    if (a.GetComputeDeviceId() != c.GetComputeDeviceId() || b.GetComputeDeviceId() != c.GetComputeDeviceId())
        RuntimeError("GPUSparseMatrix<ElemType>::TensorShuffleScaleAndAdd: All matrices must be on the same GPU");

    if (a.m_format != MatrixFormat::matrixFormatSparseCSC || b.m_format != MatrixFormat::matrixFormatSparseCSC || c.m_format != MatrixFormat::matrixFormatSparseCSC)
        NOT_IMPLEMENTED;

    // Can't distribute the operations if we need to move values across columns
    if (a.GetNumCols() != T || keepWeight != 0 || scaleFactor != 1)
        NOT_IMPLEMENTED;

    if (a.GetNumRows() != D * S * M * K)
        LogicError("GPUSparseMatrix<ElemType>::TensorShuffleScaleAndAdd: tensor dimensions and underlying matrix dimensions don't match");

    c.Resize(a.GetNumRows(), a.GetNumCols(), a.GetNumElemAllocated(), true, false);
    c.SetNzCount(a.GetNumNZElements());

    if (c.m_nz > 0)
    {
        c.PrepareDevice();
        SyncGuard syncGuard;
        CUDA_LONG N = (CUDA_LONG) c.GetNumElemAllocated();
        int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
        _tensorShuffleScaleAndAddRowSparse<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(
            reinterpret_cast<const ElemType*>(a.BufferPointer()), // source nz values
            a.RowLocation(),
            a.ColLocation(),
            reinterpret_cast<ElemType*>(c.BufferPointer()), // target nz values
            c.RowLocation(),
            c.ColLocation(),
            D, S, M, K, T,
            c.GetNumElemAllocated());
    }
    else
    {
        CUDA_CALL(hipMemset(c.BufferPointer(), 0, c.BufferSizeAllocated()));
    }
}

// backward pass from hidden layer to feature weight
// dense X sparse = sparse
template <class ElemType>
void GPUSparseMatrix<ElemType>::MultiplyAndAdd(ElemType alpha, const GPUMatrix<ElemType>& lhs, const bool transposeA,
                                               const GPUSparseMatrix<ElemType>& rhs, const bool transposeB, GPUSparseMatrix<ElemType>& c)
{
    if (!c.OwnBuffer())
        LogicError("Cannot modify since the buffer is managed externally.");

    if (lhs.GetComputeDeviceId() != rhs.GetComputeDeviceId())
        RuntimeError("GPUSparseMatrix::MultiplyAndAdd: All matrices must be on the same GPU");

    int m = transposeA ? (int) lhs.GetNumCols() : (int) lhs.GetNumRows();
    int k = transposeA ? (int) lhs.GetNumRows() : (int) lhs.GetNumCols();
    int l = transposeB ? (int) rhs.GetNumCols() : (int) rhs.GetNumRows();
    int n = transposeB ? (int) rhs.GetNumRows() : (int) rhs.GetNumCols();

    assert(m > 0 && k > 0 && l > 0 && n > 0);
    (void) m;
    (void) n; // converting from size_t to int may cause overflow
    assert(k == l);
    if (k != l)
    {
        InvalidArgument("GPUSparseMatrix::MultiplyAndAdd: The inner dimensions of a and b must match.");
    }

    if (!transposeA && !transposeB)
    {
        NOT_IMPLEMENTED;
    }
    else if (!transposeA && transposeB)
    {
        if (rhs.GetFormat() != matrixFormatSparseCSC)
            NOT_IMPLEMENTED;

        c.SetFormat(matrixFormatSparseBlockCol);

        lhs.PrepareDevice();

        int blocksPerGrid = 0;
        SyncGuard syncGuard;

        // based on the size of m_nz in rhs and numCols in the resulted matrix we use different approaches
        if (n * 10 < GridDim::maxThreadsPerBlock * rhs.m_nz)
        {
            c.Resize(m, n, 1, true, false); // reserve memory for BlockId2ColOrRow() and ColOrRow2BlockId()

            size_t* blockSize = TracingGPUMemoryAllocator::Allocate<size_t>(lhs.GetComputeDeviceId(), 1);
            CUDA_CALL(hipMemset(blockSize, 0, sizeof(size_t)));

            CUDA_CALL(hipMemset(c.BlockId2ColOrRow(), 0, sizeof(GPUSPARSE_INDEX_TYPE) * (n)));

            blocksPerGrid = (int) ceil(((double) rhs.m_nz) / GridDim::maxThreadsPerBlock);
            _findColsWithValues<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(
                rhs.RowLocation(), c.BlockId2ColOrRow(), rhs.m_nz);
                
            blocksPerGrid = (int) ceil(((double) n) / GridDim::maxThreadsPerBlock);
            _determineBlockIds<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(
                c.BlockId2ColOrRow(), c.ColOrRow2BlockId(), n, blockSize);

                
            CUDA_CALL(hipMemcpy(&c.m_blockSize, blockSize, sizeof(size_t), hipMemcpyDeviceToHost));
            TracingGPUMemoryAllocator::Free<size_t>(lhs.GetComputeDeviceId(), blockSize);

            size_t nnz = m * c.m_blockSize;
            c.Resize(m, n, nnz, true, true); // we need to keep the col2blockid and blockid2col info when resizing.
            c.m_nz = nnz;
            CUDA_CALL(hipMemset(c.BufferPointer(), 0, sizeof(ElemType) * (c.m_elemSizeAllocated)));

            LONG64 N = (LONG64) lhs.GetNumElements(); // here we process for each row in lhs and each column in rhs (==columns in lhs)
            blocksPerGrid = (int) ceil(((double) N) / GridDim::maxThreadsPerBlock);
            _denseMulSparseCSCTransposeToSparseBlockCol2<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(
                alpha,
                lhs.BufferPointer(),
                m,
                l,
                rhs.BufferPointer(),
                rhs.RowLocation(),
                rhs.ColLocation(),
                c.ColOrRow2BlockId(),
                c.BufferPointer());
        }
        else
        {
            c.m_blockSize = rhs.IdentifyRowsWithValues();
            size_t nnz = m * c.m_blockSize;
            c.Resize(m, n, nnz, true, false);
            c.m_nz = nnz;
            CUDA_CALL(hipMemset(c.BufferPointer(), 0, sizeof(ElemType) * (c.m_elemSizeAllocated)));
            CUDA_CALL(hipMemset(c.BlockId2ColOrRow(), 0, sizeof(GPUSPARSE_INDEX_TYPE) * (c.m_blockSize)));

            LONG64 N = (LONG64) lhs.GetNumElements(); // here we process for each row in lhs and each column in rhs (==columns in lhs)
            blocksPerGrid = (int) ceil(((double) N) / GridDim::maxThreadsPerBlock);
            _denseMulSparseCSCTransposeToSparseBlockCol<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock, 0, t_stream>>>(
                alpha,
                lhs.BufferPointer(),
                m,
                l,
                rhs.BufferPointer(),
                rhs.RowLocation(),
                rhs.ColLocation(),
                rhs.m_rowToId,
                c.BufferPointer(),
                c.BlockId2ColOrRow());
        }

    }
    else if (transposeA && !transposeB)
    {
        NOT_IMPLEMENTED;
    }
    else
    {
        NOT_IMPLEMENTED;
    }
}

// find the rows of rhs with values
template <class ElemType>
size_t GPUSparseMatrix<ElemType>::IdentifyRowsWithValues() const
{
    if (GetFormat() != matrixFormatSparseCSC)
        NOT_IMPLEMENTED;

    map<size_t, GPUSPARSE_INDEX_TYPE> indexer;
    GPUSPARSE_INDEX_TYPE* rowToId = (GPUSPARSE_INDEX_TYPE*) ReserveTempHostBuffer(sizeof(GPUSPARSE_INDEX_TYPE) * m_nz * 2);
    GPUSPARSE_INDEX_TYPE* h_Row = rowToId + m_nz;
    CUDA_CALL(hipMemcpy(h_Row, RowLocation(), sizeof(GPUSPARSE_INDEX_TYPE) * m_nz, hipMemcpyDeviceToHost));

    for (size_t i = 0; i < m_nz; i++)
    {
        size_t row = h_Row[i];
        if (indexer.find(row) == indexer.end())
        {
            size_t id = indexer.size(); // We need to assign size to a temp variable due to difference in Linux and Windows
            indexer[row] = id;
        }
        rowToId[i] = indexer[row];
    }
    CUDA_CALL(hipMemcpy(m_rowToId, rowToId, sizeof(GPUSPARSE_INDEX_TYPE) * m_nz, hipMemcpyHostToDevice));
    return indexer.size();
}

// used for gradients udpate
template <class ElemType>
void GPUSparseMatrix<ElemType>::ScaleAndAdd(const ElemType alpha, const GPUSparseMatrix<ElemType>& lhs, GPUMatrix<ElemType>& rhs)
{
    if (lhs.GetNumRows() != rhs.GetNumRows() || lhs.GetNumCols() != rhs.GetNumCols())
        LogicError("ScaleAndAdd: dimension mismatch");

    if (lhs.GetComputeDeviceId() != rhs.GetComputeDeviceId())
        RuntimeError("GPUSparseMatrix::ScaleAndAdd: All matrices must be on the same GPU");

    if (lhs.m_format == matrixFormatSparseBlockCol || lhs.m_format == matrixFormatSparseBlockRow)
    {
        bool blockCol = (lhs.m_format == matrixFormatSparseBlockCol);

        SyncGuard syncGuard;
        LONG64 N = (LONG64) lhs.GetNumNZElements();
        int blocksPerGrid = (int) ceil(((double) N) / GridDim::maxThreadsPerBlock);
        _scaleSparseBlockAndAddToDense<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(
            alpha,
            blockCol,
            lhs.GetNumRows(),
            lhs.GetNumCols(),
            lhs.m_blockSize,
            lhs.BufferPointer(),
            lhs.BlockId2ColOrRow(),
            rhs.BufferPointer());

    }
    else
    {
        ScaleAndAdd(alpha, lhs, 1, rhs, rhs);
    }
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceTruncate(const ElemType threshold)
{
    if (!OwnBuffer())
        LogicError("Cannot modify since the buffer is managed externally.");

    CUDA_LONG N = (CUDA_LONG) GetNumNZElements();

    CUDA_LONG blocksPerGrid = (CUDA_LONG) ceil(N * 1.0 / GridDim::maxThreadsPerBlock);
    SyncGuard syncGuard;
    ElemType* values = NzValues();
    _inplaceTruncate<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(values, threshold, N);
    return *this;
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceSoftThreshold(const ElemType threshold)
{
    if (!OwnBuffer())
        LogicError("Cannot modify since the buffer is managed externally.");

    CUDA_LONG N = (CUDA_LONG) GetNumNZElements();

    CUDA_LONG blocksPerGrid = (CUDA_LONG) ceil(N * 1.0 / GridDim::maxThreadsPerBlock);
    SyncGuard syncGuard;
    ElemType* values = NzValues();
    _inplaceSoftThreshold<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(values, threshold, N);
    return *this;
}

// normal update for smoothed gradients c and current gradients (this)
template <class ElemType>
void GPUSparseMatrix<ElemType>::NormalGrad(GPUMatrix<ElemType>& c, const ElemType momentum)
{
    if (!OwnBuffer())
        LogicError("Cannot modify since the buffer is managed externally.");

    if (c.IsEmpty())
    {
        c.Resize(GetNumRows(), GetNumCols());
        c.SetValue(0.0);
    }

    if (m_format == matrixFormatSparseBlockCol || m_format == matrixFormatSparseBlockRow)
    {
        bool isBlockCol = (m_format == MatrixFormat::matrixFormatSparseBlockCol);
        SyncGuard syncGuard;
        LONG64 N = (LONG64) GetNumNZElements();
        int blocksPerGrid = (int) ceil(((double) N) / GridDim::maxThreadsPerBlock);

        _normalGradForSparseBlock<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(
            momentum,
            isBlockCol,
            GetNumRows(),
            GetNumCols(),
            m_blockSize,
            BufferPointer(),
            BlockId2ColOrRow(),
            c.BufferPointer());
    }
    else
    {
        NOT_IMPLEMENTED;
    }
}

template <class ElemType>
ElemType GPUSparseMatrix<ElemType>::Adagrad(GPUMatrix<ElemType>& c, const bool needAveMultiplier)
{
    if (!OwnBuffer())
        LogicError("Cannot modify since the buffer is managed externally.");

    size_t numColsNeeded = GetNumCols();
    if (needAveMultiplier)
        numColsNeeded += GetNumCols();

    if (c.IsEmpty() || c.GetNumCols() < numColsNeeded)
    {
        c.Resize(GetNumRows(), numColsNeeded);
        c.SetValue(0.0);
    }

    assert(c.GetNumRows() == GetNumRows() && c.GetNumCols() == numColsNeeded);

    size_t n = this->GetNumElements();

    ElemType* multipliers = nullptr;
    if (needAveMultiplier)
        multipliers = c.GetArray() + n; // temp memory used to store multipliers,

    if (m_format == MatrixFormat::matrixFormatSparseCSC || m_format == MatrixFormat::matrixFormatSparseCSR)
    {
        NOT_IMPLEMENTED;
    }
    else if (m_format == MatrixFormat::matrixFormatSparseBlockCol || m_format == MatrixFormat::matrixFormatSparseBlockRow)
    {
        int blocksPerGrid = (m_nz + GridDim::maxThreadsPerBlock - 1) / GridDim::maxThreadsPerBlock;
        bool colMajor = (m_format == MatrixFormat::matrixFormatSparseBlockCol ? true : false);
        size_t len = colMajor ? GetNumRows() : GetNumCols();
        _adagrad4BlockSparse<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(c.GetArray(), c.GetNumRows(), BufferPointer(), BlockId2ColOrRow(), multipliers, colMajor, len, m_nz);
    }
    else
        NOT_IMPLEMENTED;

    if (!needAveMultiplier)
        return 1;

    hipblasHandle_t cuHandle = GPUMatrix<ElemType>::GetCublasHandle(GetComputeDeviceId());
    if (sizeof(ElemType) == sizeof(float))
    {
        float aveMultiplier = 0;
        CUBLAS_CALL(hipblasSasum(cuHandle, (LONG64) m_nz, reinterpret_cast<float*>(multipliers), 1, &aveMultiplier));
        return (ElemType) aveMultiplier / m_nz;
    }
    else
    {
        double aveMultiplier = 0;
        CUBLAS_CALL(hipblasDasum(cuHandle, (LONG64) m_nz, reinterpret_cast<double*>(multipliers), 1, &aveMultiplier));
        return (ElemType) aveMultiplier / m_nz;
    }
}

// sparse X dense = dense
template <class ElemType>
void GPUSparseMatrix<ElemType>::MultiplyAndWeightedAdd(ElemType alpha, const GPUSparseMatrix<ElemType>& a, const bool transposeA,
                                                       const GPUMatrix<ElemType>& b, const bool transposeD, ElemType beta, GPUMatrix<ElemType>& c)
{
    if (a.m_format != matrixFormatSparseCSR)
        NOT_IMPLEMENTED;

    if (transposeD)
        NOT_IMPLEMENTED;

    if (a.GetComputeDeviceId() != b.GetComputeDeviceId() || (b.GetComputeDeviceId() != a.GetComputeDeviceId()))
        RuntimeError("MultiplyAndWeightedAdd: All matrices must be on the same GPU");

    a.PrepareDevice();
    hipsparseHandle_t cusparseHandle = 0;
    CUSPARSE_CALL(hipsparseCreate(&cusparseHandle));
    hipsparseMatDescr_t descr = 0;
    CUSPARSE_CALL(hipsparseCreateMatDescr(&descr));
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseOperation_t oper = transposeA ? HIPSPARSE_OPERATION_TRANSPOSE : HIPSPARSE_OPERATION_NON_TRANSPOSE;

    int m = (int) a.GetNumRows();
    int n = (int) b.GetNumCols();
    assert(n == (int) c.GetNumCols());
    int k = (int) a.GetNumCols();

    SyncGuard syncGuard;
    if (sizeof(ElemType) == sizeof(float))
    {
        CUSPARSE_CALL(hipsparseScsrmm(cusparseHandle, oper, m, n, k, (int) a.GetNumElemAllocated(), reinterpret_cast<float*>(&alpha), descr, reinterpret_cast<const float*>(a.BufferPointer()),
                                     a.RowLocation(), a.ColLocation(), reinterpret_cast<float*>(b.BufferPointer()),
                                     (int) b.GetNumRows(), reinterpret_cast<float*>(&beta), reinterpret_cast<float*>(c.BufferPointer()), (int) c.GetNumRows()));
    }
    else
    {
        CUSPARSE_CALL(hipsparseDcsrmm(cusparseHandle, oper, m, n, k, (int) a.GetNumElemAllocated(), reinterpret_cast<double*>(&alpha), descr, reinterpret_cast<const double*>(a.BufferPointer()),
                                     a.RowLocation(), a.ColLocation(), reinterpret_cast<double*>(b.BufferPointer()),
                                     (int) b.GetNumRows(), reinterpret_cast<double*>(&beta), reinterpret_cast<double*>(c.BufferPointer()), (int) c.GetNumRows()));
    }
    CUSPARSE_CALL(hipsparseDestroy(cusparseHandle));
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::Multiply(const GPUSparseMatrix<ElemType>& S, const GPUMatrix<ElemType>& D, GPUMatrix<ElemType>& C)
{
    C.Resize(S.GetNumRows(), D.GetNumCols());

    MultiplyAndWeightedAdd(1, S, false, D, false, 0, C);
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::Multiply(const GPUMatrix<ElemType>& D, const GPUSparseMatrix<ElemType>& S, GPUMatrix<ElemType>& C)
{
    C.Resize(S.GetNumCols(), D.GetNumRows());

    MultiplyAndWeightedAdd(1, D, false, S, false, 0, C);
}

// ElemCountFromBufferSize - Return the elemCountAllocated for a particular buffersize
// totalBufferSize - total buffer we have to use
// return: size of allocated elements/index slots available
template <class ElemType>
size_t GPUSparseMatrix<ElemType>::ElemCountFromBufferSize(const size_t numRows, const size_t numCols, const MatrixFormat format, const size_t totalBufferSize) const
{
    size_t elemSizeAllocated;
    if (format == matrixFormatSparseCSC)
    {
        elemSizeAllocated = (totalBufferSize - sizeof(GPUSPARSE_INDEX_TYPE) * (numCols + 1)) / (sizeof(GPUSPARSE_INDEX_TYPE) + sizeof(ElemType));
    }
    else if (format == matrixFormatSparseCSR)
    {
        elemSizeAllocated = (totalBufferSize - sizeof(GPUSPARSE_INDEX_TYPE) * (numRows + 1)) / (sizeof(GPUSPARSE_INDEX_TYPE) + sizeof(ElemType));
    }
    else if (format == matrixFormatSparseBlockCol)
    {
        elemSizeAllocated = (totalBufferSize - sizeof(GPUSPARSE_INDEX_TYPE) * 2 * numCols) / sizeof(ElemType);
    }
    else if (format == matrixFormatSparseBlockCol || format == matrixFormatSparseBlockRow)
    {
        elemSizeAllocated = (totalBufferSize - sizeof(GPUSPARSE_INDEX_TYPE) * 2 * numRows) / sizeof(ElemType);
    }
    else // uncompressed COO format
    {
        elemSizeAllocated = totalBufferSize / (2 * sizeof(GPUSPARSE_INDEX_TYPE) + sizeof(ElemType));
    }
    return elemSizeAllocated;
}

template <class ElemType>
size_t GPUSparseMatrix<ElemType>::ElemCountFromBufferSize() const
{
    return ElemCountFromBufferSize(m_numRows, m_numCols, m_format, m_totalBufferSizeAllocated);
}

// PrepareBuffer - Get the dimensions start buffer, computes the starting row/column of each value
// m - rows in the source
// n - cols in the source
// canReuseBuffer - target matrix can be reused for temporary space
// func - function to call to count elements in the result (returns count, and fills csrRowPtr array)
template <class ElemType>
void GPUSparseMatrix<ElemType>::PrepareBuffer(size_t m, size_t n, bool canReuseBuffer, std::function<size_t(GPUSPARSE_INDEX_TYPE* csrRowPtrC)> func)
{
    if (!OwnBuffer())
        LogicError("Cannot modify since the buffer is managed externally.");

    if (this->m_format != matrixFormatSparseCSR)
        NOT_IMPLEMENTED;

    PrepareDevice();

    GPUSPARSE_INDEX_TYPE* csrRowPtrC = nullptr;
    GPUSparseMatrix<ElemType>& c = *this;
    size_t cSize = c.BufferSizeAllocated();
    size_t rowBufferRequired = (m + 1) * sizeof(GPUSPARSE_INDEX_TYPE);
    bool allocatedBuffer = false;

    // do we have enough memory to store just the row buffer?
    if (cSize >= rowBufferRequired && c.BufferPointer() != nullptr && canReuseBuffer)
    {
        csrRowPtrC = (GPUSPARSE_INDEX_TYPE*) c.BufferPointer();
    }
    else
    {
        csrRowPtrC = TracingGPUMemoryAllocator::Allocate<GPUSPARSE_INDEX_TYPE>(m_computeDevice, rowBufferRequired / sizeof(GPUSPARSE_INDEX_TYPE));
        allocatedBuffer = true;
    }

    // get the non-zero count from the function (and
    size_t nnzC = func(csrRowPtrC);

    // now we know the number of Non-zeros in the result set, set the output size
    c.Resize(m, n, nnzC, true, false);
    c.m_nz = nnzC;

    CUDA_CALL(hipMemcpy(c.SecondaryIndexLocation(), csrRowPtrC, c.SecondaryIndexSize(), hipMemcpyDeviceToDevice));

    // if we allocated the buffer, free it here
    if (allocatedBuffer)
        TracingGPUMemoryAllocator::Free<GPUSPARSE_INDEX_TYPE>(m_computeDevice, csrRowPtrC);
}

// Multiply - multiply one spares matrix by another sparse matrix
// S1 - first sparse matrix
// transposeS1 - transpose first matrix?
// S2 - second sparse matrix
// transposeS2 - tanspose second matrix?
// c - result matrix
// NOTE: if c has enough space allocated, it will be reused, otherwise it will be freed and a new memory block used
template <class ElemType>
void GPUSparseMatrix<ElemType>::Multiply(const GPUSparseMatrix<ElemType>& S1, bool transposeS1, const GPUSparseMatrix<ElemType>& S2, bool transposeS2, GPUSparseMatrix<ElemType>& c)
{
    if (!c.OwnBuffer())
        LogicError("Cannot modify since the buffer is managed externally.");

    if (S1.m_format != matrixFormatSparseCSR || S2.m_format != matrixFormatSparseCSR || c.m_format != matrixFormatSparseCSR)
        NOT_IMPLEMENTED;

    if (S1.GetComputeDeviceId() != S2.GetComputeDeviceId())
        RuntimeError("Sparse matrix multiply: both matrices must be on the same device");

    S1.PrepareDevice();
    hipsparseHandle_t cusparseHandle = 0;
    CUSPARSE_CALL(hipsparseCreate(&cusparseHandle));
    hipsparseMatDescr_t descrA = 0, descrB = 0, descrC = 0;
    CUSPARSE_CALL(hipsparseCreateMatDescr(&descrA));
    CUSPARSE_CALL(hipsparseCreateMatDescr(&descrB));
    CUSPARSE_CALL(hipsparseCreateMatDescr(&descrC));
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatType(descrB, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatType(descrC, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatIndexBase(descrB, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatIndexBase(descrC, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseOperation_t operA = transposeS1 ? HIPSPARSE_OPERATION_TRANSPOSE : HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipsparseOperation_t operB = transposeS2 ? HIPSPARSE_OPERATION_TRANSPOSE : HIPSPARSE_OPERATION_NON_TRANSPOSE;

    int m = int(transposeS1 ? S1.GetNumCols() : S1.GetNumRows());
    int n = int(transposeS2 ? S2.GetNumRows() : S2.GetNumCols());
    int k = int(transposeS1 ? S1.GetNumRows() : S1.GetNumCols());
    int l = int(transposeS2 ? S2.GetNumCols() : S2.GetNumRows());
    if (k != l)
        RuntimeError("Sparse matrix multiply: dimensionality mismatch");

    int nnzA = (int) S1.GetNumNZElements();
    int nnzB = (int) S2.GetNumNZElements();

    SyncGuard syncGuard;
    // Step 1
    c.PrepareBuffer(m, n, false, // false means we cannot reuse the "c" buffer if it exists for temporaries
                    [&](GPUSPARSE_INDEX_TYPE* csrRowPtrC) -> size_t
                    {
                        int nnzTotal = -1;
                        CUSPARSE_CALL(hipsparseXcsrgemmNnz(cusparseHandle, operA, operB, m, n, k, descrA, nnzA, S1.RowLocation(), S1.ColLocation(), descrB, nnzB,
                                                          S2.RowLocation(), S2.ColLocation(), descrC, csrRowPtrC, &nnzTotal));
                        return nnzTotal;
                    });

    // Step 2
    if (sizeof(float) == sizeof(ElemType))
    {
        CUSPARSE_CALL(hipsparseScsrgemm(cusparseHandle, operA, operB, m, n, k, descrA, nnzA, (const float*) S1.BufferPointer(), S1.RowLocation(), S1.ColLocation(),
                                       descrB, nnzB, (const float*) S2.BufferPointer(), S2.RowLocation(), S2.ColLocation(),
                                       descrC, (float*) c.BufferPointer(), c.RowLocation(), c.ColLocation()));
    }
    else
    {
        CUSPARSE_CALL(hipsparseDcsrgemm(cusparseHandle, operA, operB, m, n, k, descrA, nnzA, (const double*) S1.BufferPointer(), S1.RowLocation(), S1.ColLocation(),
                                       descrB, nnzB, (const double*) S2.BufferPointer(), S2.RowLocation(), S2.ColLocation(),
                                       descrC, (double*) c.BufferPointer(), c.RowLocation(), c.ColLocation()));
    }
    hipsparseDestroy(cusparseHandle);
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignProductOf(const GPUSparseMatrix<ElemType>& a, const bool transposeA, const GPUSparseMatrix<ElemType>& b, const bool transposeB)
{
    Multiply(a, transposeA, b, transposeB, *this);
    return *this;
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::ScaleAndAdd(ElemType alpha, const GPUSparseMatrix<ElemType>& a, ElemType beta, const GPUSparseMatrix<ElemType>& b, GPUSparseMatrix<ElemType>& c)
{
    if (!c.OwnBuffer())
        LogicError("Cannot modify since the buffer is managed externally.");

    if (a.m_format != matrixFormatSparseCSR || b.m_format != matrixFormatSparseCSR || c.m_format != matrixFormatSparseCSR)
    {
        NOT_IMPLEMENTED;
    }

    if (a.GetNumCols() != b.GetNumCols() || a.GetNumRows() != b.GetNumRows())
        RuntimeError("Dimensions mismatch in ScaleAndAdd");
    if (a.GetComputeDeviceId() != b.GetComputeDeviceId())
        RuntimeError("ScaleAndAdd: matrices must be on the same device");

    int m = (int) a.GetNumRows();
    int n = (int) a.GetNumCols();
    int nnzA = (int) a.GetNumNZElements();
    int nnzB = (int) b.GetNumNZElements();

    a.PrepareDevice();
    hipsparseHandle_t cusparseHandle = 0;
    CUSPARSE_CALL(hipsparseCreate(&cusparseHandle));
    hipsparseMatDescr_t descrA = 0, descrB = 0, descrC = 0;
    CUSPARSE_CALL(hipsparseCreateMatDescr(&descrA));
    CUSPARSE_CALL(hipsparseCreateMatDescr(&descrB));
    CUSPARSE_CALL(hipsparseCreateMatDescr(&descrC));
    hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatType(descrB, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatType(descrB, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatIndexBase(descrB, HIPSPARSE_INDEX_BASE_ZERO);
    hipsparseSetMatIndexBase(descrC, HIPSPARSE_INDEX_BASE_ZERO);

    SyncGuard syncGuard;
    // Step 1
    bool inOutParameter = (&b == &c);
    c.PrepareBuffer(m, n, !inOutParameter, 
                    [&](GPUSPARSE_INDEX_TYPE* csrRowPtrC) -> size_t
                    {
                        int nnzTotal = -1;
                        CUSPARSE_CALL(hipsparseXcsrgeamNnz(cusparseHandle, m, n, descrA, nnzA, a.RowLocation(), a.ColLocation(), descrB, nnzB, b.RowLocation(), b.ColLocation(), descrC, csrRowPtrC, &nnzTotal));
                        return nnzTotal;
                    });

    // Step 2
    if (sizeof(ElemType) == sizeof(float))
    {
        CUSPARSE_CALL(hipsparseScsrgeam(cusparseHandle, m, n, reinterpret_cast<const float*>(&alpha), descrA, nnzA, reinterpret_cast<const float*>(a.BufferPointer()), a.RowLocation(), a.ColLocation(),
                                       reinterpret_cast<const float*>(&beta), descrB, nnzB, reinterpret_cast<const float*>(b.BufferPointer()), b.RowLocation(), b.ColLocation(), descrC, reinterpret_cast<float*>(c.BufferPointer()), c.RowLocation(), c.ColLocation()));
    }
    else
    {
        CUSPARSE_CALL(hipsparseDcsrgeam(cusparseHandle, m, n, reinterpret_cast<const double*>(&alpha), descrA, nnzA, reinterpret_cast<const double*>(a.BufferPointer()), a.RowLocation(), a.ColLocation(),
                                       reinterpret_cast<const double*>(&beta), descrB, nnzB, reinterpret_cast<const double*>(b.BufferPointer()), b.RowLocation(), b.ColLocation(), descrC, reinterpret_cast<double*>(c.BufferPointer()), c.RowLocation(), c.ColLocation()));
    }
    hipsparseDestroy(cusparseHandle);
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::ScaleAndAdd(ElemType alpha, const GPUSparseMatrix<ElemType>& a, ElemType beta, const GPUMatrix<ElemType>& b, GPUMatrix<ElemType>& c)
{
    if (a.m_format != matrixFormatSparseCSR)
        NOT_IMPLEMENTED;

    if (a.GetNumRows() != b.GetNumRows() || a.GetNumRows() != c.GetNumRows() || a.GetNumCols() != b.GetNumCols() || a.GetNumCols() != c.GetNumCols())
        LogicError("ScaleAndAdd: dimension mismatch");
    if (a.GetComputeDeviceId() != b.GetComputeDeviceId() || a.GetComputeDeviceId() != c.GetComputeDeviceId())
        RuntimeError("ScaleAndAdd: matrices must be on the same device");
    b.PrepareDevice();
    // copy b to c
    CUDA_CALL(hipMemcpy(c.BufferPointer(), b.BufferPointer(), sizeof(ElemType) * b.GetNumElements(), hipMemcpyDeviceToDevice));
    if (beta != 1)
    {
        c *= beta;
    }
    SyncGuard syncGuard;
    CUDA_LONG M = (CUDA_LONG) a.GetNumRows();
    int blocksPerGrid = (int) ceil(1.0 * M / GridDim::maxThreadsPerBlock);
    _sparseCSRPlusDense<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(alpha, a.BufferPointer(), a.RowLocation(), a.ColLocation(), c.BufferPointer(), M);
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::ScaleAndAdd(ElemType alpha, const GPUMatrix<ElemType>& a, ElemType beta, const GPUSparseMatrix<ElemType>& b, GPUMatrix<ElemType>& c)
{
    ScaleAndAdd(beta, b, alpha, a, c);
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::Scale(ElemType alpha, GPUSparseMatrix<ElemType>& a)
{
    if (!a.OwnBuffer())
        LogicError("Cannot modify since the buffer is managed externally.");

    if (a.IsEmpty())
        return;

    CUDA_LONG N = (CUDA_LONG) a.GetNumNZElements();
    int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
    SyncGuard syncGuard;
    _scaleArray<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(alpha, a.NzValues(), N);
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::ElementWisePower(ElemType alpha, const GPUSparseMatrix<ElemType>& a, GPUSparseMatrix<ElemType>& c)
{
    if (!c.OwnBuffer())
        LogicError("Cannot modify since the buffer is managed externally.");

    if (a.GetComputeDeviceId() != c.GetComputeDeviceId())
    {
        InvalidArgument("All matrices must be on the same GPU");
    }
    else
    {
        if (a.IsEmpty())
            LogicError("ElementWisePower:  The input matrix a is empty.");

        c.ResizeAsAndCopyIndexFrom(a);

        SyncGuard syncGuard;
        a.PrepareDevice();
        CUDA_LONG N = (CUDA_LONG) a.GetNumNZElements();
        int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
        _elementWisePowerOnCuda<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(alpha, a.NzValues(), c.NzValues(), N);
    }
}

// sparse x dense = scalar
template <class ElemType>
ElemType GPUSparseMatrix<ElemType>::InnerProductOfMatrices(const GPUSparseMatrix<ElemType>& a, const GPUMatrix<ElemType>& b)
{
    if (a.m_format != matrixFormatSparseCSR && a.m_format != matrixFormatSparseCSC)
        NOT_IMPLEMENTED;

    if (a.GetComputeDeviceId() != b.GetComputeDeviceId())
        RuntimeError("a and b must be on the same device");

    int m = (int) a.GetNumRows();
    int n = (int) a.GetNumCols();
    int nnz = (int) a.GetNumNZElements();

    ElemType* cscValA = nullptr;
    GPUSPARSE_INDEX_TYPE* cscRowIndA = nullptr;
    GPUSPARSE_INDEX_TYPE* cscColPtrA = nullptr;

    hipsparseAction_t cpVals = HIPSPARSE_ACTION_NUMERIC;
    hipsparseIndexBase_t idxBase = HIPSPARSE_INDEX_BASE_ZERO;
    hipsparseHandle_t cusparseHandle = 0;

    if (a.m_format == matrixFormatSparseCSR) // need to put a in ColumnMajor format
    {
        cscValA = TracingGPUMemoryAllocator::Allocate<ElemType>(a.GetComputeDeviceId(), nnz);
        cscRowIndA = TracingGPUMemoryAllocator::Allocate<GPUSPARSE_INDEX_TYPE>(a.GetComputeDeviceId(), nnz);
        cscColPtrA = TracingGPUMemoryAllocator::Allocate<GPUSPARSE_INDEX_TYPE>(a.GetComputeDeviceId(), (n + 1));

        CUSPARSE_CALL(hipsparseCreate(&cusparseHandle));
        SyncGuard syncGuard;
        if (sizeof(ElemType) == sizeof(float))
        {
            CUSPARSE_CALL(hipsparseScsr2csc(cusparseHandle, m, n, nnz, reinterpret_cast<const float*>(a.BufferPointer()), a.RowLocation(), a.ColLocation(), reinterpret_cast<float*>(cscValA), cscRowIndA, cscColPtrA, cpVals, idxBase));
        }
        else
        {
            CUSPARSE_CALL(hipsparseDcsr2csc(cusparseHandle, m, n, nnz, reinterpret_cast<const double*>(a.BufferPointer()), a.RowLocation(), a.ColLocation(), reinterpret_cast<double*>(cscValA), cscRowIndA, cscColPtrA, cpVals, idxBase));
        }
    }
    else if (a.m_format == matrixFormatSparseCSC)
    {
        cscValA = (ElemType*) a.BufferPointer();
        cscRowIndA = a.RowLocation();
        cscColPtrA = a.ColLocation();
    }
    else
    {
        NOT_IMPLEMENTED;
    }
    // Given sparse matrix in column major format, calculate indices for corresponding sparse vector
    GPUSPARSE_INDEX_TYPE* vectArray = TracingGPUMemoryAllocator::Allocate<GPUSPARSE_INDEX_TYPE>(a.GetComputeDeviceId(), a.m_nz);
    CUDA_LONG M = n;
    CUDA_LONG N = m;
    // GPUSPARSE_INDEX_TYPE* h_vectArray= new int[a.m_nz];
    int blocksPerGrid = (int) ceil(1.0 * M / GridDim::maxThreadsPerBlock);
    SyncGuard syncGuard;
    _getSparseVectorRepresntationForCSCMatrix<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(cscColPtrA, cscRowIndA, vectArray, M, N);
    TracingGPUMemoryAllocator::Free<GPUSPARSE_INDEX_TYPE>(a.GetComputeDeviceId(), cscRowIndA);
    TracingGPUMemoryAllocator::Free<GPUSPARSE_INDEX_TYPE>(a.GetComputeDeviceId(), cscColPtrA);
    // CUDA_CALL(hipMemcpy(h_vectArray,vectArray,sizeof(GPUSPARSE_INDEX_TYPE)*a.m_nz,hipMemcpyDeviceToHost));

    // Actual dot product
    ElemType res = 0;
    if (sizeof(ElemType) == sizeof(float))
    {
        CUSPARSE_CALL(hipsparseSdoti(cusparseHandle, (int) a.m_nz, reinterpret_cast<float*>(cscValA), vectArray,
                                    reinterpret_cast<float*>(b.BufferPointer()),
                                    reinterpret_cast<float*>(&res), idxBase));
    }
    else
    {
        CUSPARSE_CALL(hipsparseDdoti(cusparseHandle, (int) a.m_nz, reinterpret_cast<double*>(cscValA), vectArray,
                                    reinterpret_cast<double*>(b.BufferPointer()),
                                    reinterpret_cast<double*>(&res), idxBase));
    }
    TracingGPUMemoryAllocator::Free<GPUSPARSE_INDEX_TYPE>(a.GetComputeDeviceId(), vectArray);
    TracingGPUMemoryAllocator::Free<ElemType>(a.GetComputeDeviceId(), cscValA);
    CUSPARSE_CALL(hipsparseDestroy(cusparseHandle));
    return res;
}

template <class ElemType>
ElemType GPUSparseMatrix<ElemType>::InnerProductOfMatrices(const GPUMatrix<ElemType>& a, const GPUSparseMatrix<ElemType>& b)
{
    return GPUSparseMatrix<ElemType>::InnerProductOfMatrices(b, a);
}

// This is an utility function useful for debugging issues with sparse matrices.
// It just checks that the CSC format indices are not corrupted / pointing to invalid memory.
template <class ElemType>
bool GPUSparseMatrix<ElemType>::IsValid() const
{
    if (m_format != MatrixFormat::matrixFormatSparseCSC)
        NOT_IMPLEMENTED;

    long* res = new long[4];
    res[0] = 1;
    res[1] = 0;
    res[2] = 0;
    res[3] = 0;
    long* d_res = TracingGPUMemoryAllocator::Allocate<long>(m_computeDevice, 4);
    CUDA_CALL(hipMemcpy(d_res, res, sizeof(long) * 4, hipMemcpyHostToDevice));

    SyncGuard syncGuard;
    int blocksPerGrid = (int) ceil((1.0 * SecondaryIndexCount()) / GridDim::maxThreadsPerBlock);
    _isValid<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(MajorIndexLocation(), SecondaryIndexLocation(), GetNumRows(), GetNumCols(), GetNumNZElements(), d_res);

    CUDA_CALL(hipMemcpy(res, d_res, sizeof(long) * 4, hipMemcpyDeviceToHost));

    if (res[0] == 1)
    {
        return true;
    }
    else
    {
        fprintf(stderr, "GPUSparseMatrix::IsValid returned false (additional info: %ld %ld %ld %ld)\n", res[0], res[1], res[2], res[3]);
        return false;
    }
}

template <class ElemType>
/*static*/ bool GPUSparseMatrix<ElemType>::AreEqual(const GPUSparseMatrix<ElemType>& a, const GPUSparseMatrix<ElemType>& b,
                                                    const ElemType threshold)
{
    if (a.GetNumNZElements() != b.GetNumNZElements() || a.GetNumRows() != b.GetNumRows() || a.GetNumCols() != b.GetNumCols())
        return false;

    if (a.m_format != b.m_format)
        NOT_IMPLEMENTED;

    long* res = new long[3];
    res[0] = 1;
    res[1] = 1;
    res[2] = 1;
    long* d_res = TracingGPUMemoryAllocator::Allocate<long>(a.GetComputeDeviceId(), 3);
    CUDA_CALL(hipMemcpy(d_res, res, sizeof(long) * 3, hipMemcpyHostToDevice));

    int blocksPerGrid = (int) ceil(1.0 * a.GetNumNZElements() / GridDim::maxThreadsPerBlock);
    _areEqual<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(a.NzValues(), b.NzValues(), (CUDA_LONG) a.GetNumNZElements(), threshold, d_res);
    _areEqual<int><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(a.ColLocation(), b.ColLocation(), (CUDA_LONG) a.GetNumNZElements(), (int) threshold, d_res + 1);
    blocksPerGrid = (int) ceil((1.0 * a.GetNumRows() + 1.0) / GridDim::maxThreadsPerBlock);
    _areEqual<int><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(a.RowLocation(), b.RowLocation(), (CUDA_LONG) a.GetNumRows() + 1, (int) threshold, d_res + 2);

    CUDA_CALL(hipMemcpy(res, d_res, sizeof(long) * 3, hipMemcpyDeviceToHost));
    if (res[0] * res[1] * res[2] == 1)
        return true;
    else
        return false;
}

template <class ElemType>
/*static*/ bool GPUSparseMatrix<ElemType>::AreEqual(const GPUMatrix<ElemType>& a, const GPUSparseMatrix<ElemType>& b,
                                                    const ElemType threshold)
{
    if (a.GetNumRows() != b.GetNumRows() || a.GetNumCols() != b.GetNumCols())
        return false;
    GPUSparseMatrix<ElemType> c(b.GetComputeDeviceId(), b.GetFormat());
    c.SetValue(a);
    return AreEqual(c, b, threshold);
}

template <class ElemType>
/*static*/ bool GPUSparseMatrix<ElemType>::AreEqual(const GPUSparseMatrix<ElemType>& a, const GPUMatrix<ElemType>& b,
                                                    const ElemType threshold)
{
    if (a.GetNumRows() != b.GetNumRows() || a.GetNumCols() != b.GetNumCols())
        return false;
    GPUSparseMatrix<ElemType> c(a.GetComputeDeviceId(), a.GetFormat());
    c.SetValue(b);
    return AreEqual(a, c, threshold);
}

template <class ElemType>
bool GPUSparseMatrix<ElemType>::IsEqualTo(const GPUSparseMatrix<ElemType>& a, const ElemType threshold) const
{
    return AreEqual(*this, a, threshold);
}

template <class ElemType>
bool GPUSparseMatrix<ElemType>::IsEqualTo(const GPUMatrix<ElemType>& a, const ElemType threshold) const
{
    return AreEqual(*this, a, threshold);
}

#pragma endregion Static BLAS Functions

#pragma region Member BLAS Functions

template <class ElemType>
DEVICEID_TYPE GPUSparseMatrix<ElemType>::GetComputeDeviceId() const
{
    // for externally managed memory the CUDA context will have the current device
    if (!OwnBuffer())
    {
        DEVICEID_TYPE devId;
        CUDA_CALL(hipGetDevice(&devId));
        return devId;
    }
    else
        return m_computeDevice;
}

// sparse x dense = dense
template <class ElemType>
GPUMatrix<ElemType> GPUSparseMatrix<ElemType>::ElementProductOf(const GPUSparseMatrix<ElemType>& a, const GPUMatrix<ElemType>& b)
{
    if (!b.OwnBuffer())
        LogicError("Cannot modify since the buffer is managed externally.");

    if (a.m_format != matrixFormatSparseCSR)
        NOT_IMPLEMENTED;

    if (a.GetNumRows() != b.GetNumRows() || a.GetNumCols() != b.GetNumCols())
        LogicError("ElementProductOf: matrix dimensions mismatch");

    b.PrepareDevice();
    GPUMatrix<ElemType> c(b.GetNumRows(), b.GetNumCols(), b.GetComputeDeviceId());

    SyncGuard syncGuard;
    CUDA_LONG M = (CUDA_LONG) a.GetNumRows();
    int blocksPerGrid = (int) ceil(1.0 * M / GridDim::maxThreadsPerBlock);
    _sparseCSRElemMulDense<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(a.BufferPointer(), a.RowLocation(), a.ColLocation(), b.BufferPointer(), c.BufferPointer(), M);
    return c;
}

// sparse x dense = dense
template <class ElemType>
GPUMatrix<ElemType> GPUSparseMatrix<ElemType>::ElementProductOf(const GPUMatrix<ElemType>& a, const GPUSparseMatrix<ElemType>& b)
{
    return GPUSparseMatrix<ElemType>::ElementProductOf(b, a);
}

template <class ElemType>
GPUSparseMatrix<ElemType> GPUSparseMatrix<ElemType>::operator+(const GPUSparseMatrix<ElemType>& a) const
{
    GPUSparseMatrix<ElemType> res(GetComputeDeviceId(), GetFormat());
    GPUSparseMatrix<ElemType>::ScaleAndAdd(1, *this, 1, a, res);
    return res;
}

template <class ElemType>
GPUSparseMatrix<ElemType> GPUSparseMatrix<ElemType>::operator-(const GPUSparseMatrix<ElemType>& a) const
{
    GPUSparseMatrix<ElemType> res(GetComputeDeviceId(), GetFormat());
    GPUSparseMatrix<ElemType>::ScaleAndAdd(1, *this, -1, a, res);
    return res;
}

// TODO: This is an unusual use of this operator. Remove this.
template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::operator^=(ElemType alpha)
{
    GPUSparseMatrix<ElemType>& us = *this;
    ElementWisePower(alpha, us, us);
    return us;
}

// TODO: This is an unusual use of this operator. Remove this.
template <class ElemType>
GPUSparseMatrix<ElemType> GPUSparseMatrix<ElemType>::operator^(ElemType alpha) const
{
    GPUSparseMatrix<ElemType> c(GetComputeDeviceId(), GetFormat());
    ElementWisePower(alpha, *this, c);
    return c;
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::operator*=(ElemType alpha)
{
    GPUSparseMatrix<ElemType>& us = *this;
    if (alpha != 1)
        Scale(alpha, us);
    return us;
}

template <class ElemType>
GPUSparseMatrix<ElemType> GPUSparseMatrix<ElemType>::operator*(ElemType alpha) const
{
    GPUSparseMatrix<ElemType> c(*this);
    if (alpha != 1)
        Scale(alpha, c);
    return c;
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignElementPowerOf(const GPUSparseMatrix<ElemType>& a, const ElemType power)
{
    ElementWisePower(power, a, *this);
    return *this;
}

template <class ElemType>
GPUSparseMatrix<ElemType> GPUSparseMatrix<ElemType>::Transpose() const
{
    int m = (int) GetNumRows();
    int n = (int) GetNumCols();
    int nnz = (int) GetNumNZElements();
    hipsparseAction_t cpVals = HIPSPARSE_ACTION_NUMERIC;
    hipsparseIndexBase_t idxBase = HIPSPARSE_INDEX_BASE_ZERO;

    assert(GetFormat() & matrixFormatCompressed); // for now this only supports compressed formats
    PrepareDevice();
    GPUSparseMatrix c(GetComputeDeviceId(), GetFormat());
    c.Resize(n, m, nnz, GetFormat(), true, false);
    c.m_nz = nnz;

    hipsparseHandle_t cusparseHandle = 0;
    CUSPARSE_CALL(hipsparseCreate(&cusparseHandle));

    SyncGuard syncGuard;
    if (m_format == MatrixFormat::matrixFormatSparseCSR)
    {
        if (nnz > 0)
        {
            if (sizeof(ElemType) == sizeof(float))
            {
                CUSPARSE_CALL(hipsparseScsr2csc(cusparseHandle, m, n, nnz, reinterpret_cast<const float*>(this->BufferPointer()), this->RowLocation(), this->ColLocation(),
                                               reinterpret_cast<float*>(c.BufferPointer()), c.ColLocation(), c.RowLocation(), cpVals, idxBase));
            }
            else
            {
                CUSPARSE_CALL(hipsparseDcsr2csc(cusparseHandle, m, n, nnz, reinterpret_cast<const double*>(this->BufferPointer()), this->RowLocation(), this->ColLocation(),
                                               reinterpret_cast<double*>(c.BufferPointer()), c.ColLocation(), c.RowLocation(), cpVals, idxBase));
            }
        }
        else
        {
            CUDA_CALL(hipMemset(c.BufferPointer(), 0, c.BufferSizeAllocated()));
        }
    }
    else if (m_format == matrixFormatSparseCSC)
    {
        if (nnz > 0)
        {
            if (sizeof(ElemType) == sizeof(float))
            {
                CUSPARSE_CALL(hipsparseScsr2csc(cusparseHandle, n, m, nnz, reinterpret_cast<const float*>(this->BufferPointer()), this->ColLocation(), this->RowLocation(),
                                               reinterpret_cast<float*>(c.BufferPointer()), c.RowLocation(), c.ColLocation(), cpVals, idxBase));
            }
            else
            {
                CUSPARSE_CALL(hipsparseDcsr2csc(cusparseHandle, n, m, nnz, reinterpret_cast<const double*>(this->BufferPointer()), this->ColLocation(), this->RowLocation(),
                                               reinterpret_cast<double*>(c.BufferPointer()), c.RowLocation(), c.ColLocation(), cpVals, idxBase));
            }
        }
        else
        {
            CUDA_CALL(hipMemset(c.BufferPointer(), 0, c.BufferSizeAllocated()));
        }
    }
    else
    {
        NOT_IMPLEMENTED;
    }
    CUSPARSE_CALL(hipsparseDestroy(cusparseHandle));
    return c;
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignTransposeOf(const GPUSparseMatrix<ElemType>& a)
{
    if (!OwnBuffer())
        LogicError("Cannot modify since the buffer is managed externally.");

    if (this == &a)
        LogicError("AssignTransposeOf: a is the same as [this]. Does not support inplace transpose.");

    if (a.IsEmpty())
        LogicError("AssignTransposeOf: Matrix a is empty.");

    *this = a.Transpose();
    return *this;
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::InplaceTranspose()
{
    if (IsEmpty())
        return;
    // transfer converted block over to this pointer
    *this = std::move(Transpose());
}

template <class ElemType>
GPUSparseMatrix<ElemType> GPUSparseMatrix<ElemType>::ColumnSlice(size_t startColumn, size_t numCols) const
{
    if (startColumn + numCols > m_numCols)
        InvalidArgument("The slice (%d+%d) is out of range of the source matrix (%d).", (int) startColumn, (int) numCols, (int) m_numCols);

    if (m_format != MatrixFormat::matrixFormatSparseCSC && (startColumn != 0 || numCols != m_numCols))
        NOT_IMPLEMENTED;

    GPUSparseMatrix<ElemType> slice(m_computeDevice);
    slice.m_computeDevice            = m_computeDevice;
    slice.m_numRows                  = m_numRows;
    slice.m_numCols                  = numCols;
    slice.m_nz                       = (numCols == m_numCols) ? m_nz : SecondaryIndexValueAt(startColumn + numCols) - SecondaryIndexValueAt(startColumn);
    slice.m_elemSizeAllocated        = m_elemSizeAllocated;
    slice.m_totalBufferSizeAllocated = m_totalBufferSizeAllocated;
    slice.m_pArray                   = m_pArray;
    slice.m_format                   = m_format;
    slice.m_externalBuffer           = true;
    slice.m_sliceOf                  = const_cast<GPUSparseMatrix<ElemType>*>(this); // BUGBUG: ColumnSlice() returns a reference to a mutable matrix, even if itself is 'const'; should not be.
    slice.m_blockSize                = m_blockSize;
    slice.m_rowToId                  = m_rowToId;
    slice.m_tempHostBuffer           = m_tempHostBuffer;
    slice.m_tempHostBufferSize       = m_tempHostBufferSize;
    slice.m_sliceViewOffset          = startColumn; // Just shift the compressed index location to the new startColumn - that's it!

    return slice;
}

template <class ElemType>
GPUMatrix<ElemType> GPUSparseMatrix<ElemType>::CopyColumnSliceToDense(size_t startColumn, size_t numCols) const
{
    int m = (int) GetNumRows();
    int n = (int) GetNumCols();

    // if (numCols == 0)
    //    LogicError("The slice cannot have 0 columns.");

    if (startColumn + numCols > n)
        InvalidArgument("The slice (%d+%d) is out of range of the source matrix (%d).", (int) startColumn, (int) numCols, (int) n);

    if (m_format != MatrixFormat::matrixFormatSparseCSC)
        NOT_IMPLEMENTED;

    GPUMatrix<ElemType> slice(m, numCols, GetComputeDeviceId());

    PrepareDevice();
    hipsparseHandle_t cusparseHandle = 0;
    CUSPARSE_CALL(hipsparseCreate(&cusparseHandle));
    hipsparseMatDescr_t descr = 0;
    CUSPARSE_CALL(hipsparseCreateMatDescr(&descr));
    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

    SyncGuard syncGuard;
    CUSPARSE_CALL(hipsparseSetStream(cusparseHandle, t_stream));
    if (sizeof(ElemType) == sizeof(float))
    {
        CUSPARSE_CALL(hipsparseScsc2dense(cusparseHandle, m, numCols, descr, (float*) BufferPointer(), RowLocation(), ColLocation() + startColumn, (float*) slice.BufferPointer(), m));
    }
    else
    {
        CUSPARSE_CALL(hipsparseDcsc2dense(cusparseHandle, m, numCols, descr, (double*) BufferPointer(), RowLocation(), ColLocation() + startColumn, (double*) slice.BufferPointer(), m));
    }

    CUSPARSE_CALL(hipsparseDestroy(cusparseHandle));

    return slice;
}

template <class ElemType>
GPUMatrix<ElemType> GPUSparseMatrix<ElemType>::DiagonalToDense() const
{
    int m = (int) GetNumRows();
    int n = (int) GetNumCols();

    if (m != n)
        LogicError("Diagonal can be called only for square matrix. (rows=%d, cols=%d)", m, n);

    if (m_format != MatrixFormat::matrixFormatSparseCSC)
        NOT_IMPLEMENTED;

    GPUMatrix<ElemType> tmp(m, n, GetComputeDeviceId());

    // TODO: Implement optimized diagonal functions for sparse matrices. For now copy to dense first.
    CopyToDenseMatrix(tmp);

    return tmp.Diagonal();
}

template <class ElemType>
ElemType GPUSparseMatrix<ElemType>::SumOfAbsElements() const
{
    if (IsEmpty())
        return 0;

    hipblasHandle_t cuHandle = GPUMatrix<ElemType>::GetCublasHandle(GetComputeDeviceId());
    if (sizeof(ElemType) == sizeof(float))
    {
        float res = 0;
        hipblasSasum(cuHandle, (int) GetNumNZElements(), reinterpret_cast<const float*>(NzValues()), 1, &res);
        return res;
    }
    else
    {
        double res = 0;
        hipblasDasum(cuHandle, (int) GetNumNZElements(), reinterpret_cast<const double*>(NzValues()), 1, &res);
        return ElemType(res);
    }
}

template <class ElemType>
ElemType GPUSparseMatrix<ElemType>::SumOfElements() const
{
    if (IsEmpty())
        LogicError("SumOfElements: Matrix is empty");

    ElemType* d_sum = TracingGPUMemoryAllocator::Allocate<ElemType>(m_computeDevice, 1);
    ElemType h_sum;
    // WARNING: THIS kernel is not the most efficient way!
    _reductionSum<ElemType><<<1, 1024>>>(NzValues(), d_sum, (LONG64) GetNumNZElements());
    CUDA_CALL(hipMemcpy(&h_sum, d_sum, sizeof(ElemType), hipMemcpyDeviceToHost));
    TracingGPUMemoryAllocator::Free<ElemType>(m_computeDevice, d_sum);

    return h_sum;
}

// sqrt(sum all elements^2)
template <class ElemType>
ElemType GPUSparseMatrix<ElemType>::FrobeniusNorm() const
{
    if (IsEmpty())
        return 0;

    ElemType* d_sum = TracingGPUMemoryAllocator::Allocate<ElemType>(m_computeDevice, 1);
    ElemType h_sum = 0;
    // WARNING: THIS kernel is not the most efficient way!
    _reductionSum2<ElemType><<<1, 1024>>>(NzValues(), d_sum, (int) GetNumNZElements());
    CUDA_CALL(hipMemcpy(&h_sum, d_sum, sizeof(ElemType), hipMemcpyDeviceToHost));
    TracingGPUMemoryAllocator::Free<ElemType>(m_computeDevice, d_sum);

    if (sizeof(ElemType) == sizeof(float))
        return (ElemType) sqrtf((float) h_sum);
    else
        return (ElemType) sqrt((double) h_sum);
}

template <class ElemType>
ElemType GPUSparseMatrix<ElemType>::MatrixNormInf() const
{
    if (IsEmpty())
        return 0;

    ElemType* d_maxAbs = TracingGPUMemoryAllocator::Allocate<ElemType>(m_computeDevice, 1);
    ElemType h_maxAbs = 0;
    // WARNING: THIS kernel is not the most efficient way!
    _reductionMatrixNormInf<ElemType><<<1, 1024>>>(NzValues(), d_maxAbs, (int) GetNumNZElements());
    CUDA_CALL(hipMemcpy(&h_maxAbs, d_maxAbs, sizeof(ElemType), hipMemcpyDeviceToHost));
    TracingGPUMemoryAllocator::Free<ElemType>(m_computeDevice, d_maxAbs);

    if (sizeof(ElemType) == sizeof(float))
        return h_maxAbs;
    else
        return h_maxAbs;
}

template <class ElemType>
ElemType GPUSparseMatrix<ElemType>::MatrixNorm1() const
{
    return SumOfAbsElements();
}

#pragma endregion Member BLAS Functions

#pragma region Other Functions

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::ElementInverse()
{
#if 1
    // Note: This makes no sense because sparse matrices are defined by having lots of zeroes.
    NOT_IMPLEMENTED;
#else
    if (!OwnBuffer())
        LogicError("Cannot modify since the buffer is managed externally.");

    if (IsEmpty())
        LogicError("ElementInverse: Matrix is empty.");

    CUDA_LONG N = (CUDA_LONG) GetNumNZElements();
    int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
    SyncGuard syncGuard;
    _elemInverse<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(NzValues(), N);
    return *this;
#endif
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignElementInverseOf(const GPUSparseMatrix<ElemType>& a)
{
#if 1
    // Note: This makes no sense because sparse matrices are defined by having lots of zeroes.
    UNUSED(a); NOT_IMPLEMENTED;
#else
    SetValue(a);
    return ElementInverse();
#endif
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceSigmoid()
{
#if 1
    // Note: This makes no sense because sigmoid(0) != 0.
    NOT_IMPLEMENTED;
#else
    performElementWiseFunction(ElementWiseOperator::opSigmoid, *this);
    return *this;
#endif
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignSigmoidOf(const GPUSparseMatrix<ElemType>& a)
{
#if 1
    // Note: This makes no sense because sigmoid(0) != 0.
    UNUSED(a); NOT_IMPLEMENTED;
#else
    if (this != &a)
        Resize(a.GetNumRows(), a.GetNumCols());
    performElementWiseFunction(ElementWiseOperator::opSigmoid, a);
    return *this;
#endif
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceLinearRectifierDerivative()
{
    performElementWiseFunction(ElementWiseOperator::opLinearRectifierDerivative, *this);
    return *this;
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignLinearRectifierDerivativeOf(const GPUSparseMatrix<ElemType>& a)
{
    if (this != &a)
        Resize(a.GetNumRows(), a.GetNumCols());
    performElementWiseFunction(ElementWiseOperator::opLinearRectifierDerivative, a);
    return *this;
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceTanh()
{
    performElementWiseFunction(ElementWiseOperator::opTanh, *this);
    return *this;
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignTanhOf(const GPUSparseMatrix<ElemType>& a)
{
    if (this != &a)
        Resize(a.GetNumRows(), a.GetNumCols());
    performElementWiseFunction(ElementWiseOperator::opTanh, a);
    return *this;
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceSqrt()
{
    performElementWiseFunction(ElementWiseOperator::opSqrt, *this);
    return *this;
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignSqrtOf(const GPUSparseMatrix<ElemType>& a)
{
    if (this != &a)
        Resize(a.GetNumRows(), a.GetNumCols());
    performElementWiseFunction(ElementWiseOperator::opSqrt, a);
    return *this;
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceExp()
{
#if 1
    // Note: This makes no sense because exp(0) != 0.
    NOT_IMPLEMENTED;
#else
    performElementWiseFunction(ElementWiseOperator::opExp, *this);
    return *this;
#endif
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignExpOf(const GPUSparseMatrix<ElemType>& a)
{
#if 1
    // Note: This makes no sense because exp(0) != 0.
    UNUSED(a); NOT_IMPLEMENTED;
#else
    if (this != &a)
        Resize(a.GetNumRows(), a.GetNumCols());
    performElementWiseFunction(ElementWiseOperator::opExp, a);
    return *this;
#endif
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceLog()
{
#if 1
    // Note: This makes no sense because log(0) != 0.
    NOT_IMPLEMENTED;
#else
    performElementWiseFunction(ElementWiseOperator::opLog, *this);
    return *this;
#endif
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignLogOf(const GPUSparseMatrix<ElemType>& a)
{
#if 1
    // Note: This makes no sense because log(0) != 0.
    UNUSED(a); NOT_IMPLEMENTED;
#else
    if (this != &a)
        Resize(a.GetNumRows(), a.GetNumCols());
    performElementWiseFunction(ElementWiseOperator::opLog, a);
    return *this;
#endif
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceAbs()
{
    performElementWiseFunction(ElementWiseOperator::opAbs, *this);
    return *this;
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignAbsOf(const GPUSparseMatrix<ElemType>& a)
{
    if (this != &a)
        Resize(a.GetNumRows(), a.GetNumCols());
    performElementWiseFunction(ElementWiseOperator::opAbs, a);
    return *this;
}

// TODO: Check whether these functions always map 0 to 0.
template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceTruncateBottom(const ElemType threshold)
{
    if (!OwnBuffer())
        LogicError("Cannot modify since the buffer is managed externally.");

    if (IsEmpty())
        LogicError("InplaceTruncateBottom: Matrix is empty.");
    CUDA_LONG N = (CUDA_LONG) GetNumNZElements();
    int blocksPerGrid = (int) ceil(N * 1.0 / GridDim::maxThreadsPerBlock);
    SyncGuard syncGuard;
    _assignTruncateBottom<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(NzValues(), NzValues(), threshold, N);
    return *this;
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignTruncateBottomOf(const GPUSparseMatrix<ElemType>& a, const ElemType threshold)
{
    if (!OwnBuffer())
        LogicError("Cannot modify since the buffer is managed externally.");

    if (a.IsEmpty())
        LogicError("AssignTruncateBottomOf: Matrix a is empty.");

    if (this != &a)
    {
        // Resize(a.GetNumRows(), a.GetNumCols());
        ResizeAsAndCopyIndexFrom(a);
    }
    CUDA_LONG N = (CUDA_LONG) GetNumNZElements();
    int blocksPerGrid = (int) ceil(N * 1.0 / GridDim::maxThreadsPerBlock);
    SyncGuard syncGuard;
    _assignTruncateBottom<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(NzValues(), a.NzValues(), threshold, N);
    return *this;
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceTruncateTop(const ElemType threshold)
{
    if (!OwnBuffer())
        LogicError("Cannot modify since the buffer is managed externally.");

    if (IsEmpty())
        LogicError("InplaceTruncateTop: Matrix is empty.");
    CUDA_LONG N = (CUDA_LONG) GetNumNZElements();
    int blocksPerGrid = (int) ceil(N * 1.0 / GridDim::maxThreadsPerBlock);
    SyncGuard syncGuard;
    _assignTruncateTop<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(NzValues(), NzValues(), threshold, N);
    return *this;
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignTruncateTopOf(const GPUSparseMatrix<ElemType>& a, const ElemType threshold)
{
    if (!OwnBuffer())
        LogicError("Cannot modify since the buffer is managed externally.");

    if (a.IsEmpty())
        LogicError("AssignTruncateTopOf: Matrix a is empty.");

    if (this != &a)
    {
        ResizeAsAndCopyIndexFrom(a);
    }

    CUDA_LONG N = (CUDA_LONG) GetNumNZElements();
    int blocksPerGrid = (int) ceil(N * 1.0 / GridDim::maxThreadsPerBlock);
    SyncGuard syncGuard;
    _assignTruncateTop<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(NzValues(), a.NzValues(), threshold, N);
    return *this;
}

template <class ElemType>
GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::SetToZeroIfAbsLessThan(const ElemType threshold)
{
    if (!OwnBuffer())
        LogicError("Cannot modify since the buffer is managed externally.");

    if (IsEmpty())
        LogicError("SetToZeroIfAbsLessThan: Matrix is empty.");
    CUDA_LONG N = (CUDA_LONG) GetNumNZElements();
    int blocksPerGrid = (int) ceil(N * 1.0 / GridDim::maxThreadsPerBlock);
    SyncGuard syncGuard;
    _setToZeroIfAbsLessThan<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(NzValues(), threshold, N);
    return *this;
}

#pragma endregion

#pragma region Helper Functions

//outBuffer should be allocated to be >= size by the caller
template <class ElemType>
template <class OutType, class InType>
/*private*/ void GPUSparseMatrix<ElemType>::CopyBuffer(OutType* outBuffer, const InType* inBuffer, const size_t size)
{
#pragma omp parallel for
    for (size_t i = 0; i < (size & ~3); i += 4)
    {
        outBuffer[i] = inBuffer[i];
        outBuffer[i + 1] = inBuffer[i + 1];
        outBuffer[i + 2] = inBuffer[i + 2];
        outBuffer[i + 3] = inBuffer[i + 3];
    }
    // handle remaining stuffs
    for (size_t i = size & ~3; i < size; i++)
    {
        outBuffer[i] = inBuffer[i];
    }
}

template <class ElemType>
void* GPUSparseMatrix<ElemType>::ReserveTempHostBuffer(const size_t sizeInByte) const
{
    if (m_tempHostBufferSize < sizeInByte)
    {
        delete[](byte*) m_tempHostBuffer;
        m_tempHostBuffer = new byte[sizeInByte];
        m_tempHostBufferSize = sizeInByte;
    }
    return (void*) m_tempHostBuffer;
}

template <class ElemType>
void GPUSparseMatrix<ElemType>::performElementWiseFunction(ElementWiseOperator kind, const GPUSparseMatrix<ElemType>& src)
{
    if (!OwnBuffer())
        LogicError("Cannot modify since the buffer is managed externally.");

    CUDA_LONG N = (CUDA_LONG) GetNumNZElements();
    int blocksPerGrid = (int) ceil(1.0 * N / GridDim::maxThreadsPerBlock);
    SyncGuard syncGuard;
    switch (kind)
    {
    case ElementWiseOperator::opSigmoid:
        return _elementWiseSigmoidOnCuda<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(src.NzValues(), NzValues(), N);
    case ElementWiseOperator::opTanh:
        return _elementWiseTanhOnCuda<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(src.NzValues(), NzValues(), N);
    case ElementWiseOperator::opSqrt:
        return _elementWiseSqrtOnCuda<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(src.NzValues(), NzValues(), N);
    case ElementWiseOperator::opExp:
        return _elementWiseExpOnCuda<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(src.NzValues(), NzValues(), N);
    case ElementWiseOperator::opLog:
        return _elementWiseLogOnCuda<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(src.NzValues(), NzValues(), N);
    case ElementWiseOperator::opAbs:
        return _elementWiseAbsOnCuda<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(src.NzValues(), NzValues(), N);
    case ElementWiseOperator::opLinearRectifierDerivative:
        return _elementWiseLinRectDerivativeOnCuda<ElemType><<<blocksPerGrid, GridDim::maxThreadsPerBlock>>>(src.NzValues(), NzValues(), N);
    default:
        NOT_IMPLEMENTED;
    }
}

#pragma endregion Helper Functions

template class MATH_API GPUSparseMatrix<float>;
template class MATH_API GPUSparseMatrix<double>;

// We use Matrix<char> as the backing store for QuantizedMatrix
// Let's explicitly instantiate the methods we need for that purpose
template GPUSparseMatrix<char>::GPUSparseMatrix(DEVICEID_TYPE computeDevice, const MatrixFormat matrixFormat);
template GPUSparseMatrix<char>::GPUSparseMatrix(const size_t numRows, const size_t numCols, const size_t numNZ, DEVICEID_TYPE computeDevice, const MatrixFormat matrixFormat);
template GPUSparseMatrix<char>::GPUSparseMatrix(GPUSparseMatrix<char> const&);
template GPUSparseMatrix<char>::GPUSparseMatrix(GPUSparseMatrix<char>&&);
template void GPUSparseMatrix<char>::SetValue(CPUSparseMatrix<char> const&);
template void GPUSparseMatrix<char>::SetValue(GPUSparseMatrix<char> const&);
template void GPUSparseMatrix<char>::SetValue(GPUMatrix<char> const&);
template void GPUSparseMatrix<char>::CopyToDenseMatrix(GPUMatrix<char>&) const;
template void GPUSparseMatrix<char>::CopyToCPUSparseMatrix(CPUSparseMatrix<char>&) const;
template void GPUSparseMatrix<char>::ChangeDeviceTo(int);
template void GPUSparseMatrix<char>::Resize(const size_t numRows, const size_t numCols, const size_t numNZElemToReserve, const bool growOnly, bool keepExistingValues);
template void GPUSparseMatrix<char>::Reset();
template GPUSparseMatrix<char>::~GPUSparseMatrix();
template GPUSparseMatrix<char> GPUSparseMatrix<char>::ColumnSlice(size_t startColumn, size_t numCols) const;
template GPUMatrix<char> GPUSparseMatrix<char>::CopyColumnSliceToDense(size_t startColumn, size_t numCols) const;
template GPUSparseMatrix<char>& GPUSparseMatrix<char>::operator=(GPUSparseMatrix<char>&& deepCopy);

template <class ElemType>
MATH_API File& operator>>(File& stream, GPUSparseMatrix<ElemType>& us)
{
    if (!us.OwnBuffer())
        LogicError("Cannot read into a managed external matrix");

    stream.GetMarker(fileMarkerBeginSection, std::wstring(L"BMAT"));
    size_t elsize;
    stream >> elsize;
    if (sizeof(ElemType) != elsize)
        RuntimeError("Template argument size doesn't match those in file");
    std::wstring matrixName;

    // now prepare this header to receive the data being read
    size_t nz, colnum, rownum;
    int format;

    // read in the header information
    stream >> matrixName >> format >> nz >> colnum >> rownum;

    us.m_format = (MatrixFormat) format;
    if (us.m_format != matrixFormatSparseCSC && us.m_format != matrixFormatSparseCSR)
        NOT_IMPLEMENTED;

    us.Resize(rownum, colnum, nz, true, false);
    us.SetNzCount(nz);

    if (nz > 0)
    {
        size_t compressedSize = (us.m_format == matrixFormatSparseCSC) ? colnum + 1 : rownum + 1;
        ElemType* dataBuffer = new ElemType[nz];
        CPUSPARSE_INDEX_TYPE* unCompressedIndex = new CPUSPARSE_INDEX_TYPE[nz];
        CPUSPARSE_INDEX_TYPE* compressedIndex = new CPUSPARSE_INDEX_TYPE[compressedSize];

        // read in the sparse matrix info
        for (size_t i = 0; i < nz; ++i)
        {
            stream >> dataBuffer[i];
        }
        for (size_t i = 0; i < nz; ++i)
        {
            size_t val;
            stream >> val;
            unCompressedIndex[i] = val;
        }
        for (size_t i = 0; i < compressedSize; ++i)
        {
            size_t val;
            stream >> val;
            compressedIndex[i] = val;
        }

        if (us.m_format == matrixFormatSparseCSC)
            us.SetMatrixFromCSCFormat(compressedIndex, unCompressedIndex, dataBuffer, nz, rownum, colnum);
        else if (us.m_format == matrixFormatSparseCSR)
            us.SetMatrixFromCSRFormat(compressedIndex, unCompressedIndex, dataBuffer, nz, rownum, colnum);

        delete[] dataBuffer;
        delete[] unCompressedIndex;
        delete[] compressedIndex;
    }

    stream.GetMarker(fileMarkerEndSection, std::wstring(L"EMAT"));

    return stream;
}

template MATH_API File& operator>>(File& stream, GPUSparseMatrix<float>& us);
template MATH_API File& operator>>(File& stream, GPUSparseMatrix<double>& us);

template <class ElemType>
MATH_API File& operator<<(File& stream, const GPUSparseMatrix<ElemType>& us)
{
    if (us.m_format != matrixFormatSparseCSC && us.m_format != matrixFormatSparseCSR)
        NOT_IMPLEMENTED;

    stream.PutMarker(fileMarkerBeginSection, std::wstring(L"BMAT"));
    stream << sizeof(ElemType);
	std::wstring s(L"nnmatrix");
	stream << s;

    size_t nz = us.GetNumNZElements(), numElemAllocated = us.GetNumElemAllocated(), numRows = us.GetNumRows(), numCols = us.GetNumCols();
    size_t compressedSize = us.SecondaryIndexCount();
    int format = us.GetFormat();

    stream << format << nz << numCols << numRows;

    if (nz > 0)
    {
        ElemType* dataBuffer = nullptr;
        CPUSPARSE_INDEX_TYPE* compressedIndex = nullptr;
        CPUSPARSE_INDEX_TYPE* unCompressedIndex = nullptr;

        if (us.m_format == matrixFormatSparseCSC)
            us.GetMatrixFromCSCFormat(compressedIndex, unCompressedIndex, dataBuffer, numElemAllocated, nz, numRows, numCols);
        else if (us.m_format == matrixFormatSparseCSR)
            us.GetMatrixFromCSRFormat(compressedIndex, unCompressedIndex, dataBuffer, numElemAllocated, nz, numRows, numCols);
        else
            NOT_IMPLEMENTED;

        for (size_t i = 0; i < nz; ++i)
        {
            stream << dataBuffer[i];
        }
        for (size_t i = 0; i < nz; ++i)
        {
            size_t val = unCompressedIndex[i];
            stream << val;
        }
        for (size_t i = 0; i < compressedSize; ++i)
        {
            size_t val = compressedIndex[i];
            stream << val;
        }

        delete[] dataBuffer;
        delete[] unCompressedIndex;
        delete[] compressedIndex;
    }

    stream.PutMarker(fileMarkerEndSection, std::wstring(L"EMAT"));

    return stream;
}

template MATH_API File& operator<<(File& stream, const GPUSparseMatrix<float>& us);
template MATH_API File& operator<<(File& stream, const GPUSparseMatrix<double>& us);

}}}

#endif // CPUONLY
